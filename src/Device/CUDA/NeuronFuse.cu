#include "hip/hip_runtime.h"

// #include "../ggex/GG_util.hpp"       //ugly  "__builtin_ia32_ldtilecfg" is undefined
#include "./cuda_common.h"
#include "./cublas_common.h"
#include "./llm_c/matmul.cuh"
#include "./llm_c/layernorm.cuh"
#include "./llm_c/encoder.cuh"
#include "./llm_c/fused_classifier.cuh"
// #include "./TE/fused_attn/fused_attn_fp8.cu"
#include "../../Manifold/Neuron.hpp"
#include "../../Manifold/Fish.hpp"
#include "./EDevice.hpp"
// #include "./mfu.h"
#define NOMINMAX

hipblasComputeType_t cublas_compute = HIPBLAS_COMPUTE_32F;
const size_t cublaslt_workspace_size = 32 * 1024 * 1024;
hipblasLtHandle_t cublaslt_handle;
void* cublaslt_workspace = NULL;
hipStream_t main_stream=nullptr;
hipDeviceProp_t deviceProp;


int EDGE_DEVICES::GPU_::MAX_COUNT = 16;     //  16    
std::vector<EDGE_DEVICES::GPU_> EDGE_DEVICES::GPU_::hipGetDevice(int flag) {
    std::vector<GPU_> devices;
#ifdef __HIP_PLATFORM_AMD__
    // Workaround for a rocBLAS bug when using multiple graphics cards:
    // https://github.com/ROCmSoftwarePlatform/rocBLAS/issues/1346
    {
        int major_version = 0;
        size_t version_length = 0;
        if (rocblas_get_version_string_size(&version_length) == rocblas_status_success) {
            std::vector<char> version(version_length+1, '\0');
            if (rocblas_get_version_string(version.data(), version.size()) == rocblas_status_success) {
                version.resize(::strlen(version.data()));
                int parsed_value = 0;
                if (std::from_chars(version.data(), version.data() + version.size(), parsed_value).ec == std::errc()) {
                    major_version = parsed_value;
                }
            }
        }
        if (major_version < 4) {
            _LOG_DEBUG(_CUDA_NAME " calling rocblas_initialize as a workaround for a rocBLAS bug\n");
            rocblas_initialize();
            CUDA_CHECK(hipDeviceSynchronize());
        }
    }
#endif
    int device_count = 0;
    hipError_t err = hipGetDeviceCount(&device_count);    //CUDA functions do not throw exceptions, why?
    if (err != hipSuccess) {
        _ERROR("%s: failed to initialize CUDA: %s\n", __func__, hipGetErrorString(err));
        return devices;
    }
    // assert(device_count <= GPU_DEVICE::MAX_COUNT);

    int64_t total_vram = 0;
#ifdef _CUDA_FORCE_MMQ
    _INFO("%s: _CUDA_FORCE_MMQ:    yes\n", __func__);
#else
    _INFO("%s: _CUDA_FORCE_MMQ:    no\n", __func__);
#endif // _CUDA_FORCE_MMQ
#ifdef _CUDA_FORCE_CUBLAS
    _INFO("%s: _CUDA_FORCE_CUBLAS: yes\n", __func__);
#else
    _INFO("%s: _CUDA_FORCE_CUBLAS: no\n", __func__);
#endif // _CUDA_FORCE_CUBLAS
    _INFO("%s: found %d CUDA devices:\n", __func__, device_count);
    devices.resize(device_count);
    for (int id = 0; id < device_count; ++id) {
        int device_vmm = 0;

#if defined(_USE_VMM)
        hipDevice_t device;
        CU_CHECK(hipDeviceGet(&device, id));
        CU_CHECK(hipDeviceGetAttribute(&device_vmm, hipDeviceAttributeVirtualMemoryManagementSupported, device));

        if (device_vmm) {
            hipMemAllocationProp alloc_prop = {};
            alloc_prop.type = hipMemAllocationTypePinned;
            alloc_prop.location.type = hipMemLocationTypeDevice;
            alloc_prop.location.id = id;
            CU_CHECK(hipMemGetAllocationGranularity(&devices[id].vmm_granularity, &alloc_prop, hipMemAllocationGranularityRecommended));
        }
#endif // defined(_USE_VMM)
        devices[id].vmm = !!device_vmm;
        hipDeviceProp_t prop;
        cudaCheck(hipGetDeviceProperties(&prop, id));
        total_vram += prop.totalGlobalMem;
        devices[id].nsm       = prop.multiProcessorCount;
        devices[id].smpb      = prop.sharedMemPerBlock;
        devices[id].warp_size = prop.warpSize;
#if defined(_USE_HIP) && defined(__HIP_PLATFORM_AMD__)
        devices[id].smpbo = prop.sharedMemPerBlock;

        devices[id].cc = ggml_cuda_parse_id(prop.gcnArchName);
        if ((devices[id].cc & 0xff00) == 0x0) {
            _LOG_WARN("invalid architecture ID received for device %d %s: %s  cc %d.%d\n",
                            id, prop.name, prop.gcnArchName, prop.major, prop.minor);

            // Fallback to prop.major and prop.minor
            if (prop.major > 0) {
                devices[id].cc = _CUDA_CC_OFFSET_AMD + prop.major * 0x100;
                devices[id].cc += prop.minor * 0x10;
            }
        }
        _INFO("  Device %d: %s, %s (0x%x), VMM: %s, Wave Size: %d\n",
                        id, prop.name, prop.gcnArchName, devices[id].cc & 0xffff,
                        device_vmm ? "yes" : "no", prop.warpSize);
#elif defined(_USE_MUSA)
        // TODO: refine the .cc to reflect MUSA's actual CC capabilities
        devices[id].smpbo = prop.sharedMemPerBlockOptin;
        devices[id].cc = 100*prop.major + 10*prop.minor;
        _INFO("  Device %d: %s, compute capability %d.%d, VMM: %s\n",
                        id, prop.name, prop.major, prop.minor, device_vmm ? "yes" : "no");
#else
        devices[id].smpbo = prop.sharedMemPerBlockOptin;
        devices[id].cc = 100*prop.major + 10*prop.minor;
        _INFO("  Device %d: %s, compute capability %d.%d, VMM: %s\n",
                        id, prop.name, prop.major, prop.minor, device_vmm ? "yes" : "no");
#endif // defined(_USE_HIP) && defined(__HIP_PLATFORM_AMD__)
    }

    return devices;
}



hGTensor huTensor::_Multiply(const hGTensor& b) {
    huTensor *cuB=dynamic_cast<huTensor *>(b.get());
    assert(cuB!=nullptr);
    return nullptr;
}

bool TokenEmbed::UpdateBucket(int type,int flag){
    num_c_groups = CEIL_DIV(C, (WARP_SIZE * x128::size));
    if (bucket_info != NULL)
        return false;
    
    assert((size_t)(B * T) * num_c_groups < (1ULL<<31ULL)); // todo - maybe an issue for llama3-400B(?)
    workload_indices = new int[B * T * num_c_groups];
    bucket_info = new int4[B * T * num_c_groups];
    return true;
}
void TokenEmbed::WorkloadOnBucker(int *inputs_cpu,int flag ){
    // if(num_buckets>0) 
    //     return;

    // Step 1: Sort inputs into buckets
    int total_items = 0;
    std::unordered_map<uint64_t, std::vector<uint64_t>> buckets;
    for (uint64_t bt = 0; bt < B * T; bt++) {
        for (uint64_t c_group = 0; c_group < num_c_groups; c_group++) {
            // todo - passing c_group/inputs_cpu[bt] in data to avoid a second hash lookup is a bit hacky
            uint64_t data = bt + (c_group<<32ULL) + ((uint64_t)inputs_cpu[bt]<<42ULL);
            buckets[c_group + num_c_groups * inputs_cpu[bt]].push_back(data);
            total_items++;
        }
    }
    // Step 2: Sort buckets by size in descending order
    // this is so the largest buckets are processed first by the GPU
    // otherwise, if they started late, they would still be running with the rest of the GPU idle
    std::vector<std::pair<uint64_t, std::vector<uint64_t>>> sortedBuckets(buckets.begin(), buckets.end());
    std::sort(sortedBuckets.begin(), sortedBuckets.end(), // ugly because we don't have a typedef for the std::pair
              [](const std::pair<uint64_t, std::vector<uint64_t>>& a, const std::pair<uint64_t, std::vector<uint64_t>>& b) {
                  return a.second.size() > b.second.size();
              });

    num_buckets = buckets.size();
    int bucket_index = 0;
    int workload_index = 0;
    for (const auto& bucket : sortedBuckets) {
        bucket_info[bucket_index].x = workload_index; // bucket start
        bucket_info[bucket_index].y = bucket.second.size(); // bucket size
        bucket_info[bucket_index].z = (bucket.second[0] >> 42ULL) & ((1ULL<<20ULL)-1); // bucket ix
        bucket_info[bucket_index].w = (bucket.second[0] >> 32ULL) & ((1ULL<<10ULL)-1); // bucket c

        for (uint64_t idx : bucket.second) {
            workload_indices[workload_index++] = (int)(idx & ((1ULL<<31ULL)-1ULL));
        }
        bucket_index++;
    }

    floatX *scratch=(floatX *)GTensor::buff;
    // Step 3: Copy data from host to device (async until the last one to avoid synchronising CPU/GPU twice)
    // todo - could use CUDA events (even without streams) to avoid CPU/GPU synchronisation completely
    int4* d_bucket_info = (int4*)scratch;
    int*  d_workload_indices = (int*)(scratch + B*T*num_c_groups * sizeof(int4));
    cudaCheck(hipMemcpyAsync(d_bucket_info, bucket_info, num_buckets * sizeof(int4), hipMemcpyHostToDevice, main_stream));
    cudaCheck(hipMemcpyAsync(d_workload_indices, workload_indices, total_items * sizeof(int), hipMemcpyHostToDevice, main_stream));
}


hGTensor TokenEmbed::OnEmbed(const int* tokens, int seed){
try{
    int OC=w->ne[1],Vp=padded_nCls;  
    hGTensor cur = out;
    if(isForward()){
        grid_size = CEIL_DIV(B*T*C, block_size);
        // encoder_forward(ToX(cur), tokens, ToX(w), ToX0(b), B, T, C, main_stream);
        CU_embed_forw_<<<grid_size, block_size, 0, main_stream>>>(ToX(cur), tokens, ToX(w), ToX0(b), B, T, C);
        w->Print("wte",0,0);        //ToX(w),true,Vp,C
        PrintTensor<floatX>("wpe",ToX0(b),true,T,C);
        PrintTensor<int>("inputs",tokens,true,B,T);            PrintTensor<floatX>("GetRow",ToX(cur),true,B,T,C);
        if(maec!=nullptr){
            maec->ENC(cur);
        }
    }else{        
        UpdateBucket(0x0);
        WorkloadOnBucker(hBatch->host,0x0);
        floatX *scratchX=(floatX *)GTensor::buff;
        hGTensor delta =GTensor::delta,cur=delta;  
        if(maec!=nullptr){
            cur = maec->ENC(cur);
        }     
        // encoder_backward_1(ToG(w), ToG0(b), ToX(cur), tokens, B, T, C, seed, main_stream); 
            encoder_backward(ToG(w), ToG0(b), scratchX, workload_indices, bucket_info, ToX(cur), tokens, hBatch->host, B, T, C, seed, main_stream);
            
    // PrintTensor<floatX>("grad of wte",grads.wte,true,Vp,C);         PrintTensor<float>("losses",acts.losses,true,B,T);
    // PrintTensor<floatX>("grad of wpe",grads.wpe,true,T,C);
    }
    return cur;
}catch(...){
    assert(0);
    return nullptr;
}
}

//  seed - use stochastic rounding to go from FP32 to BF16
hGTensor TokenEmbed::SubW(hGTensor hSamp,bool isForw,  hGTensor wOut, int flag){
try{
    int nSamp = hSamp->size(),*samps=TO<int>(hSamp),nLayer=hFish->config.nLayer();
    int OC=w->ne[1],Vp=padded_nCls,seed=42,T=nSamp,B=1;  
    grid_size = CEIL_DIV(B*T*C, block_size);
    hGTensor cur = wOut,wSrc=flag==0 ? w : wInv;
    
    if(isForw){
        // encoder_forward(ToX(cur), samps, ToX(wSrc), nullptr, 1, T, C, main_stream);
        CU_embed_forw_<<<grid_size, block_size, 0, main_stream>>>(ToX(cur), samps, ToX(wSrc), T, C,Vp,flag==1);    
        cur->Print("subW",0,0);     
    }else{             
        CU_embed_back_<<<grid_size, block_size, 0, main_stream>>>(ToG(wSrc), samps, ToX(cur), T, C,Vp,1.0,flag==1);    
        // encoder_backward(ToG(wSrc), nullptr, scratchX, workload_indices, bucket_info, ToX(cur), samps, hBatch->host, 1, T, C, seed, main_stream);     
    }
    return cur;
}catch(...){
    assert(0);
    return nullptr;
}
}
int SLP::Forw(hGTensor rhs_0,hGTensor lhs_,hGTensor gelu,int flag){
try{
    floatX *rhs=ToX(rhs_0),*pre_gelu = ToX0(gelu),*wX=ToX(w);//,*inp=ToX(lhs_);
    int OC=nOut,IC=nIn;
    // assert(C==w->ne[0]);
    assert(rhs_0->size()>=B*T*OC);        //  ne of scatch
    float* dbias_buffer=nullptr;
    inp = lhs_;
    bool transAW = true;
    // if(isTransW)        
    //     transAW = false;
    // matmul_forward_cublaslt(rhs, inp, wX, ToX0(b), B, T, C, OC, main_stream,pre_gelu,gelu_fusion);
    if(compression==SAMPLE && subw!=nullptr){
        subw->SubW(hSamps,true,GTensor::tmpW,samp_type);
        wX = ToX(GTensor::tmpW);        //assert(nSample==OC || nSample==IC);
        // GTensor::tmpW->Print("subW",0,-1);    
        // encoder_forward(wX, samples, ToX(w), nullptr, 1, nSample, C, main_stream);
    }
    if (gelu_fusion < 1 && pre_gelu) {
        matmul_cublaslt(pre_gelu, wX, ToX(lhs_), ToX0(b), OC, B*T, IC, main_stream, transAW, false, 0, 0, 0, 0, false, NULL, false);
        gelu_forward(rhs, pre_gelu, B*T*OC, main_stream);
    } else {
        matmul_cublaslt(rhs, wX, ToX(lhs_), ToX0(b), OC, B*T, IC, main_stream, transAW, false, 0, 0, 0, 0, false, pre_gelu, false);
    }
    if(compression==SAMPLE) {
        // rhs_0->Print(rhs_0->name,0,-1);
    }
        // PrintTensor<floatX>("l_qkvw",l_qkvw,true,3*C,C);       PrintTensor<floatX>("l_qkvb",l_qkvb,true,3*C,1);
        // PrintTensor<floatX>("l_qkvr",l_qkvr,true,B,T,3*C);
    
    return 0x0;
}catch(...){
    assert(0);
    return -1;
}
}
int SLP::Back(hGTensor delta,hGTensor inp,hGTensor deltaIn,hGTensor gelu,float* dbias_buffer,int flag){
try{
    floatX *pre_gelu = ToX0(gelu),*wX=ToX(w),*gW=ToG(w);
    int OC=nOut,IC=nIn;     
    assert(delta!=nullptr);
    deltaIn->Print("delta_in",0,flag);
    if(compression==SAMPLE && subw!=nullptr){    //remater to get wX
        subw->SubW(hSamps,true,GTensor::tmpW,samp_type);
        wX = ToX(GTensor::tmpW);        //assert(nSample==OC || nSample==IC);
        gW = ToX(GTensor::tmpGW);       
        cudaCheck(hipMemsetAsync(gW, 0, GTensor::tmpGW->nByte(), main_stream));
    }
    matmul_backward(ToX(delta), gW, ToG0(b),ToX(deltaIn),ToX(inp), wX, dbias_buffer, B, T, IC, OC, main_stream,isTransW, pre_gelu);
    if(compression==SAMPLE && subw!=nullptr){
        subw->SubW(hSamps,false,GTensor::tmpGW,samp_type);
    } 
    delta->Print("delta",0,flag);
    return 0x0;
}catch(...){
    assert(0);
    return -1;
}
}
int SLP::FUSE_cuda_block(hGTensor rhs,hGTensor lhs,hGTensor gelu,bool isForw,int flag){
    return 0x0;
}

//  hIn = QKV->out
hGTensor FFN::FUSE_cuda(hGTensor hIn,floatX *scratch,int flag){
    floatX *ff2=ToX(down.out),*ff1=ToX(up.out);
    // gelu just inplace operation on ff1, maybe could share memory!    
    hGTensor tGelu=GTensor::tmpFF1;    
    tGelu = GTensor::scratch;
    bool isBias = up.b!=nullptr;  
    
    if(isForward()){  
        if(fuseNorm==nullptr){
            norm.FUSE_cuda(hIn);       
        }
        floatX * inp1_ = ToX(norm.out);         
        if(remater_ffn)  {
            input_1 = inp1_;
            ff1=ToX(GTensor::tmpFF1);              
        } 
        assert(ff1!=nullptr);       // ff1=gelu_forward(out, l_fch_gelu, B*T*OC, stream);
        floatX *scratch = ToX(GTensor::delta);    
        if(!gate.Empty()){
            gate.Forw(tGelu,norm.out,remater_ffn?GTensor::tmpFF1:up.out);        
        }
        up.Forw(tGelu,norm.out,remater_ffn?GTensor::tmpFF1:up.out);        
        // PrintTensor<floatX>("inp1",ToX(norm.out),true,B,T,C,1,-1);          PrintTensor<floatX>("ff1",ff1,true,B,T,latent,1,-1);  
        down.Forw(GTensor::delta,tGelu,nullptr,isSymmetric);       
        // PrintTensor<floatX>("inp1",ToX(norm.out),true,B,T,C,1,-1);
        PrintTensor<floatX>("ffn",scratch,true,B,T,C);

        // fused_residual_forward5(ToX(out), normed,mean,rstd, ToX(hIn), scratch, ToX(fuseNorm->w), xb, B*T, C, main_stream);
        residual_forward(ToX(out), ToX(hIn), scratch, B*T*C, main_stream);
        if(fuseNorm!=nullptr){
            return fuseNorm->FUSE_cuda(out);   
            // layernorm_forward(ToX(fuseNorm->out), TO<float>(fuseNorm->mean),TO<float>(fuseNorm->rstd), ToX(out),ToX(fuseNorm->w), ToX0(fuseNorm->b), B*T, 1, C, main_stream);
            // return fuseNorm->out;
        }
        
        // PrintTensor<floatX>("inp1",ToX(norm.out),true,B,T,C,1,-1);
        out->Print("residual3",0,0);
    }else{
        assert(delta!=nullptr);
        // floatX *dl_bt4c = ToX(GTensor::bt4c),*dresidual = ToX(GTensor::delta),*gNb=norm.b==nullptr?nullptr:ToG(norm.b); 
        float*  scratchF = (float*) scratch;   // not the same inp1 of forward !!!
        if(input_1!=nullptr){
            input_1 =  ToX(norm.out);
            ff1=ToX(GTensor::tmpFF1);  
            up.Forw(tGelu,norm.out,GTensor::tmpFF1);            
            // fuMM(l_fch_gelu,input_1, (floatX*)up.w->data, ToX0(up.b), B, T, C, latent, main_stream, ff1, gelu_fusion);
            // norm.out->Print("inp1",0,-1);          PrintTensor<floatX>("ff1",ff1,true,B,T,latent,-1);  
        }else
            gelu_forward(ToX(tGelu), ff1, B*T*latent, main_stream);  
        assert(ff1!=nullptr);   
        down.Back(GTensor::bt4c,tGelu,GTensor::delta,GTensor::tmpFF1,scratchF);
        // matmul_backward(dl_bt4c, ToG(down.w), ToG0(down.b), dresidual, ToX(tGelu), ToX(down.w), scratchF, B, T, latent, C, main_stream, ff1, gelu_fusion);
        // PrintTensor<floatX>("back of ffn1",dl_bt4c,true,B,T,latent);
        up.Back(delta,norm.out,GTensor::bt4c,nullptr,scratchF);
        // matmul_backward(ToX(delta), ToG(up.w), ToG0(up.b), dl_bt4c, ToX(norm.out), ToX(up.w), scratchF, B, T, C, latent, main_stream);
        // // layernorm backward does += to the dresidual, so it correctly accumulates grad from the MLP block above
        // norm.FUSE_cuda(residual,scratchF,tmpDelta);
        float *_mean = norm.mean==nullptr?nullptr : TO<float>(norm.mean);
        layernorm_backward(ToX(GTensor::delta), ToG(norm.w), ToG0(norm.b), scratchF, ToX(delta), ToX(hIn), ToX(norm.w), _mean, TO<float>(norm.rstd), B, T, C, main_stream);
        // lastQKV->proj_cat.Back(delta,lastQKV->attn,GTensor::delta,nullptr,scratchF);        // matmul_backward(ToX(delta), ToG(lastQKV->proj_cat.w), ToG0(lastQKV->proj_cat.b), ToX(GTensor::delta), ToX(lastQKV->attn), ToX(lastQKV->proj_cat.w), scratchF, B, T, C, C, main_stream);
        delta->Print("back of ffn0",0,0);
    }
    
    return out;
}

/*
    layernorm_forward(floatX* out, float* mean, float* rstd, floatX* inp, const floatX* weight, const floatX* bias,         int B, int T, int C, hipStream_t stream)
    layernorm_backwar(floatX* dinp, floatX* dweight, floatX* dbias, float* scratch,const floatX* dout, const floatX* inp, const floatX* weight, const float* mean, const float* rstd,          int B, int T, int C, hipStream_t stream)
*/
hGTensor huTensor::Normal(hGTensor hOut,hGTensor _mean,hGTensor _rstd,hGTensor w,hGTensor b,bool isForward,int flag) {
    assert(!hOut->isEmpty());
    int B=hOut->ne[0],T=hOut->ne[1],C=w->ne[0];
    // assert(b!=nullptr);     
    floatX *weight=(floatX*)(w->data),*bias=ToX0(b);    //b==nullptr?nullptr:(floatX*)(b->data);    
    floatX *out=(floatX*)(hOut->data); // (B, T, C)
    if(isForward)
        layernorm_forward(out, (float*)_mean->data, (float*)_rstd->data, (floatX *)data,weight,bias, B, T, C, main_stream);
    else{
        layernorm_backward(nullptr, (floatX*)(w->grad), ToG0(b), nullptr, nullptr,nullptr, weight, 
            (float*)_mean->data, (float*)_rstd->data, B, T, C, main_stream);
    }
    
    return hOut;
}

hGTensor LayerNormal::FUSE_cuda(hGTensor inpL,float* scratch,hGTensor deltaIn,int flag) {
    float* _mean = mean==nullptr ? nullptr : TO<float>(mean);
    if(isForward()){    //cur = cur->Normal(out,mean,rstd,w,b); 
        inp = inpL;  
        layernorm_forward(ToX(out), _mean,  TO<float>(rstd), ToX(inpL),ToX(w),ToX0(b), B, T, C, main_stream);
    }        
    else{   
        assert(deltaIn!=nullptr);       // const floatX* deltaIn=ToX(GTensor::bt4c);
        // floatX* dresidual = ToX(GTensor::delta);
        layernorm_backward(ToX(delta), ToG(w), ToG0(b), scratch, ToX(deltaIn),ToX(inpL), ToX(w), _mean,  TO<float>(rstd), B, T, C, main_stream);
        delta->Print("back of normal",0,0);
    }
    return out;
}

//void fused_classifier(Type* logits, float* cuLoss,const float dloss, const int* targets,int B, int T, int V, int P, std::bool_constant<WriteDLogits> write_dlogits, hipStream_t stream) {
//float huTensor::FusedLoss(float dloss,hGTensor hLoss,hGTensor hTarget,hGTensor hLastLayer, hGTensor w,int V,bool isForward,int flag){
hGTensor OutCLS::FUSE_cuda(hGTensor inp,int flag)   {
    int V=nCls,Vp=padded_nCls, gelu_fusion=1;
    assert(proj.b==nullptr);
    mean_loss = 0.0f;
    const int *targets = (int*)(target->data);
    float* cuLoss = (float*)out->data;  
    hGTensor cur = preLogits,w = proj.w;  //==nullptr?token_embed:proj.w;    
    if(isForward()){        
        if(maec!=nullptr){
            inp = maec->DEC(inp,true);   C = inp->ne[2];
        }   
        floatX *z0=ToX(inp),*pre_gelu=nullptr;  //* errLogits = ToX(preLogits),
        cudaCheck(hipMemset(cuLoss, 0, B*T*sizeof(float)));
        assert( target->isSameShape(out) );
        constexpr std::bool_constant<true> cuFalse;    
        for(size_t i=0;i<B;i+=dB){
            size_t off=i*T*Vp,n1=i*T,nZ=i*T*C;
            off=0;      //reduce memory            
            // fuMM(ToX(cur)+off, z0+nZ, ToX(w), NULL, dB, T, C, Vp, main_stream);  //[32,1024,50304]=[32,1024,768]*[768,50304]
            matmul_cublaslt(ToX(cur)+off, ToX(w), z0+nZ, NULL, Vp, dB*T, C, main_stream, true, false, 0, 0, 0, 0, false);
            fused_classifier(ToX(cur)+off, cuLoss+n1, rLoss, targets+n1, dB, T, V, Vp, cuFalse, main_stream);        //target=[32,1024]
            if(ToG0(w)!=nullptr && delta!=nullptr){
                matmul_cublaslt(ToX(delta)+nZ, ToX(w), ToX(cur)+off, NULL, C, dB*T, Vp, main_stream, false, false, 0, 0, 0, 0, false,gelu_fusion >= 2 ? pre_gelu : NULL, true);   
                matmul_cublaslt(ToG(w), z0+nZ, ToX(cur)+off, NULL /*dbias*/, C, Vp, dB*T, main_stream, false, true, 0, 0, 0, 0,true /* accumulate */, NULL, true);                
            }                         
        }
        // fused_classifier(errLogits, cuLoss, rLoss, targets, B, T, V, Vp, cuFalse, main_stream);        //target=[32,1024]
        cudaCheck(hipMemcpy(hostLoss, cuLoss, B * T * sizeof(float), hipMemcpyDeviceToHost));                 
        cudaCheck(hipDeviceSynchronize());
        w->Print("oucls.proj.w",1,-1);
         
        /*if(flag==0x1001 && gw!=nullptr && errOut!=nullptr){            //matmul_backward(errOut, gw, NULL, errLogits, z0, w, NULL, B, T, C, Vp, main_stream);      //accumulate=true  
            matmul_cublaslt(errOut, w, errLogits, NULL, C, B*T, Vp, main_stream, false, false, 0, 0, 0, 0, false,gelu_fusion >= 2 ? pre_gelu : NULL, true);
            if (gelu_fusion < 2 && pre_gelu) {
                gelu_backward_inplace(errOut, pre_gelu, B*T*C, main_stream);
            }
            matmul_cublaslt(gw, z0, errLogits, NULL , C, Vp, B*T, main_stream, false, true, 0, 0, 0, 0,true , NULL, true);
        }*/
            
        for (int i = 0; i < B*T; i++) {
            assert(!std::isnan(hostLoss[i]));
            mean_loss += hostLoss[i];
        }   
        mean_loss /= B*T;
    }else{        
        // matmul_backward(errOut, gw, NULL, errLogits, z0, w, NULL, B, T, C, Vp, main_stream);
        if(maec!=nullptr){
            cur = maec->DEC(delta,false);
            return cur;
        }  else
            return delta;
    }
    cudaCheck(hipGetLastError());
    return preLogits;
}

huTensor::~huTensor()  {
    Free();

}

