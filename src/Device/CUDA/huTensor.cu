#include "hip/hip_runtime.h"
// #include <hip/hip_fp16.h>
// #include <hip/hip_fp8.h>
#include "../../Manifold/Fish.hpp"
#include "../../Tensor/GTensor.hpp"
#include "../../Utils/GST_log.hpp"
#include "../../Utils/GST_rander.hpp"
#include "../../g_float.hpp"
#include "./kernel/Operator.cuh"
#include "./kernel/utils.cuh"
// const int block_512 = 512;
huTensor::huTensor(Fish* fish, const string& name_, const SHAPE shape, typNUMBER tpD_, bool isAlloc, int flag) : GTensor(fish, shape, tpD_, false, flag) {
    size_t nEle = size();
    if (DEBUG.T_cpu == 1) {
        flags |= BIT_FLAG::F_HOSTALLOC;
    } else
        flags |= BIT_FLAG::F_GPU;
    // hFish->InitGensor(nullptr,name,attn,false);
    if (!name_.empty())
        snprintf(name, sizeof(name), "%s", name_.c_str());
    else
        name[0] = '\0';

    if (isAlloc) {
        Alloc(0x0, flag);
    }
}

size_t GTensor::szMaloc = 0;
size_t huTensor::mostMemory(int typ) const {
    if (BIT_TEST(flags, F_NOALLOC))
        return 0x0;
    // if(BIT_TEST(flags,F_HOSTALLOC))
    //     return 0x0;
    if (hRef != nullptr) {
        return 0x0;
    }
    size_t most = nByte();
    if (isParam() && hFish->isTrain()) {
        most += nByte();                     // grad
        most += sizeof(float) * size() * 2;  // gm,gv is float array
    }
    if (isParam()) {
        most += sizeof(float) * ne[0];
    }
    return most;
}
/*
    hipHostAlloc is a function used to allocate pinned (page-locked) host memory, which can improve data transfer performance between the host (CPU) and device
   (GPU). Pinned memory allows for faster transfers because it bypasses the operating system's virtual memory system.
*/
size_t huTensor::Alloc_1(void** dst, bool isZero, size_t sz0, int flag) {
    assert(*dst == nullptr);

    bool hostAlloc    = BIT_TEST(flags, F_HOSTALLOC);
    hipError_t error = hipSuccess;
    size_t szAlloc    = sz0 == 0 ? szData : sz0;
    assert(szAlloc > 0);
    error = hostAlloc ? hipHostAlloc(dst, szAlloc, 0) : hipMalloc(dst, szAlloc);  // 8420
    // strange behavior of callo
    // data = calloc(szAlloc,1);  sAlloc = "Alloc_c/cu";   //8386
    if (error != hipSuccess) {
        printf("[CUDA Alloc] failed @%s, ERR=%s!\n", name, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    if (isZero)
        cudaCheck(hipMemset(*dst, 0, szAlloc));
    szMaloc += szAlloc;
    return szAlloc;
}
size_t huTensor::Free_1(void** obj, const string& info) {
    assert(*obj != nullptr);
    // _INFO("\t%s%s freed@%p(%.3gM)!",name,info.c_str(),*obj,(szData)/1.0e6);
    if (BIT_TEST(flags, F_HOSTALLOC))
        hipHostFree(*obj);
    else {
        // cudaFreeCheck(obj);
        hipError_t error = hipFree(*obj);
        if (error != hipSuccess) {
            // _INFO("[CUDA ERROR] at file %s:%d:\n%s\n", file, line, hipGetErrorString(error));
            _INFO("[CUDA] free failed @\"%s\"! err=%s.\n", name, hipGetErrorString(error));
            // exit(EXIT_FAILURE);
        }
        *obj = nullptr;
    }

    *obj = nullptr;
    szMaloc -= szData;

    return szMaloc;
}

static mt19937_state rngOfParams;
bool huTensor::InitParam(int tpX) {
    size_t nElem0 = size(), i;
    size_t nInit = size(1), nB = BPE(type);
    bool isTmp = true;
    if (tpInit > 0 && tpInit != SERIALIZE) {
        // _INFO("[InitParam]\t%ld-%ld@%s\n",size(),nInit,name);

        floatX* tmp = new floatX[nInit];
        switch (tpInit) {
            case FIX_1:
                for (i = 0; i < nInit; i++) tmp[i] = (floatX)(1.0);
                break;
            default:
#ifdef NDEBUG
            {
                CU_normal<floatX>(nInit, (floatX*)data, 0.02f * residual_scale);
                isTmp = false;
            }
#else
                // manual_seed(&rngOfParams, 42);     //cys   only for debug
                // float *tmp32 = new float[nInit];
                assert(nInit < INT_MAX);
                normal_19937<floatX>(tmp, nInit, 0.0f, 0.02f * residual_scale, &rngOfParams);
                // for(i=0;i<nInit;i++)        tmp[i]=Float2T<floatX>(tmp32+i);
                // delete[] tmp32;
#endif
            break;
        }
        if (isTmp) {
            cudaCheck(hipMemcpy(data, tmp, nInit * nB, hipMemcpyHostToDevice));
        }
        delete[] tmp;
        // Print(name,0,-1);
    } else {
        if (tpInit == SERIALIZE) {  //  ???
            if (host_data != nullptr) {
                SerialGP(host_data, nullptr, szData, false);
            }
        }
    }

    return true;
}

/*
   Only for gguf-serialize
*/
bool huTensor::CopyGG(struct ggml_tensor* gg_, int flag) {
#ifdef __USE_GGML__
    int i = 0;
    assert(gg == nullptr);
    bool isAlloc = data != nullptr;
    void* src    = gg_->data;
    if (!isAlloc) {
        memcpy(name, gg_->name, sizeof(char) * GGML_MAX_NAME);
        for (i = 0; i < GGML_MAX_DIMS; i++) {
            shape.push_back(gg_->ne[i]);
            nb[i] = gg_->nb[i];
        }
        type = (typNUMBER)gg_->type;
        Alloc();
        // flags = gg_->flags;     //bug in ggml: don't support flag serialization
        double fnB = BPE(type);  // ggml_row_size  ???
        szData     = size() * fnB;
    } else {
        for (i = 0; i < shape.size(); i++) {
            if (BIT_TEST(flags, F_PADDED))
                assert(shape[i] >= gg_->ne[i]);
            else
                assert(shape[i] == gg_->ne[i]);
            if (type == (typNUMBER)gg_->type)
                assert(nb[i] == gg_->nb[i]);
        }
    }
    size_t szSrc = ggml_nbytes(gg_);
    if (type == (typNUMBER)gg_->type) {
        if (szSrc != szData) {
            if (BIT_TEST(flags, F_PADDED)) {
                assert(strcmp(name, "token_embd.weight") == 0 && szSrc <= szData);
            } else {
                assert(0);
            }
        }
    };

#ifdef _TENSOR_G_
    bool toDevice = SerialGP(src, nullptr, szSrc, false, 0x0);
    assert(toDevice);
#endif
#endif
    // if(src!=data)       delete[] src;
    return true;
}

//  From:   https://stackoverflow.com/questions/57948643/whats-a-good-way-to-zero-out-cudamallocd-data
/*__global__ void clear_scratch_space_kernel(int * data, int blocks, int threads) {
    // BOZO: change the code to just error out if we're any of the border cases below
    const int idx = blockIdx.x * threads + threadIdx.x;
    long size = sizeof(int) * COUNT;
    long size_of_typical_chunk = round_up(size / (blocks * threads), GPU_CACHE_LINE_SIZE_IN_BYTES);
    // Due to truncation, the threads at the end won't have anything to do.  This is a little sloppy but costs us
    // hardly anything in performance, so we do the simpler thing.

    long this_threads_offset = idx * size_of_typical_chunk;
    if (this_threads_offset > SIZE_OF_DATA) {
        return;
    }

    long size_of_this_threads_chunk;
    if (this_threads_offset + size_of_typical_chunk >= SIZE_OF_DATA) {
        // We are the last thread, so we do a partial write
        size_of_this_threads_chunk = SIZE_OF_DATA - this_threads_offset;
    } else {
        size_of_this_threads_chunk = size_of_typical_chunk;
    }
    void * starting_address = reinterpret_cast<void *>(reinterpret_cast<char *>(data) + this_threads_offset);
    memset((void *) starting_address, 0, size_of_this_threads_chunk);
}
__global__ void clear_scratch_space_with_coalesced_writes_kernel(int * data, int blocks, int threads) {
    if (COUNT % (blocks * threads) != 0) {
        printf("Adjust the SIZE_OF_DATA so it's divisible by the number of (blocks * threads)\n");
    }
    const long count_of_ints_in_each_blocks_chunk = COUNT / blocks;

    int block = blockIdx.x;
    int thread = threadIdx.x;

    const long rounds_needed = count_of_ints_in_each_blocks_chunk / threads;

    const long this_blocks_starting_offset = block * count_of_ints_in_each_blocks_chunk;

    //printf("Clearing %li ints starting at offset %li\n", count_of_ints_in_each_blocks_chunk, this_blocks_starting_offset);

    int * this_threads_base_pointer = &data[this_blocks_starting_offset + thread];
    for (int round = 0; round < rounds_needed; ++round) {
        *this_threads_base_pointer = 0;
        this_threads_base_pointer += threads;
    }
}
void set_gpu_data_to_ones(int * data_on_gpu) {
    hipMemset(data_on_gpu, 1, SIZE_OF_DATA);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
}
void check_gpu_data_is_zeroes(int * data_on_gpu, char * data_on_cpu) {
    hipMemcpy(data_on_cpu, data_on_gpu, SIZE_OF_DATA, hipMemcpyDeviceToHost);
    for (long i = 0; i < SIZE_OF_DATA; ++i) {
        if (data_on_cpu[i] != 0) {
            printf("Failed to zero-out byte offset %i in the data\n", i);
        }
    }
}*/

void huTensor::Zero() {
    assert(data != nullptr);
    //  https://stackoverflow.com/questions/57948643/whats-a-good-way-to-zero-out-cudamallocd-data
    cudaCheck(hipMemset(data, 0, szData));
    if (grad != nullptr) {
        ZeroGrad();
    }
}
void huTensor::ZeroGrad() {
    assert(grad != nullptr);
    cudaCheck(hipMemset(grad, 0, szData));
    // cudaCheck(hipMemsetAsync(ToG(tensor), 0, tensor->nByte(), main_stream));
}
bool cuClearGrad(std::vector<hGTensor> tensors, int flag) {
    for (auto tensor : tensors) {
        if (tensor->isRefer())
            continue;
        cudaCheck(hipMemsetAsync(ToG(tensor), 0, tensor->nByte(), main_stream));
    }

    return true;
}

bool D2H(void* dev, void* host, size_t szData, int flag) {
    try {
        assert(host != nullptr && dev != nullptr);
        cudaCheck(hipMemcpy(host, dev, szData, hipMemcpyDeviceToHost));
        return true;
    } catch (...) {
        return false;
    }
}
bool H2D(void* dev, void* host, size_t szData, int flag) {
    try {
        assert(host != nullptr && dev != nullptr);
        cudaCheck(hipMemcpy(host, dev, szData, hipMemcpyHostToDevice));
        return true;
    } catch (...) {
        return false;
    }
}

bool huTensor::SerialData(const string& info, void* host, bool isToHost, int flag) {
    try {
        if (host == nullptr) {
            assert(host_data != nullptr);
            host = host_data;
        }
        if (isToHost) {
            // cudaCheck(hipMemcpyAsync(host,data, szData, hipMemcpyDeviceToHost));
            cudaCheck(hipMemcpy(host, data, szData, hipMemcpyDeviceToHost));
        } else {
            // cudaCheck(hipMemcpyAsync(data, host,szData, hipMemcpyHostToDevice));
            cudaCheck(hipMemcpy(data, host, szData, hipMemcpyHostToDevice));
        }
        if (flag < 0) {
            char buf[1024];
            sprintf(buf, "%s:%s@%s", info.c_str(), isToHost ? "SAVE" : "LOAD", name);
            Print(buf, 0, -1);
        }

        return true;
    } catch (...) {
        return false;
    }
}
//  this <=> Y
bool huTensor::SerialGP(void* yD, void* yG, size_t szY, bool isToY, int flag) {
    try {
        if (isToY) {
            assert(szY >= szData);
            cudaCheck(hipMemcpy(yD, data, szY, hipMemcpyDeviceToHost));
            if (yG != nullptr) {
                assert(grad != nullptr);
                cudaCheck(hipMemcpy(yG, grad, szY, hipMemcpyDeviceToHost));
            }
        } else {
            assert(szY <= szData);
            cudaCheck(hipMemcpy(data, yD, szY, hipMemcpyHostToDevice));
            if (yG != nullptr) {
                assert(grad != nullptr);
                cudaCheck(hipMemcpy(grad, yG, szY, hipMemcpyHostToDevice));
                cudaCheck(hipMemcpy(grad, yG, szY, hipMemcpyHostToDevice));
            }
        }
        return true;
    } catch (...) {
        return false;
    }
}

bool huTensor::OverWrite(hGTensor hGT, bool isSrc, int flag) {
    size_t nEle = size();
    assert(isSameShape(hGT) && szData > 0);
    if (isSrc) {
        huTensor* src = dynamic_cast<huTensor*>(hGT.get());
        if (src == nullptr)  //  hGT => this
            cudaCheck(hipMemcpy(data, hGT->data, szData, hipMemcpyHostToDevice));
        else {
            cudaCheck(hipMemcpy(data, hGT->data, szData, hipMemcpyDeviceToDevice));
        }
    } else {  //  this => hGT
        assert(0);
    }

    return true;
}

hGTensor huTensor::CrossEntropy(const hGTensor b, int flag) { return b; }

// Helper function determines the maximum number of block sums
inline int get_max_num_block_sums(int* num_slices_all, int numel) {
    // NOTE: this needs to be kept in sync with `global_norm_squared` below.
    const int block_size = 512;
    const int grid_size  = deviceProp.maxThreadsPerMultiProcessor * deviceProp.multiProcessorCount / block_size;
    assert(grid_size > 0);
    int max_num_block_sums = 0;
    for (int i = 0; i < numel; i++) {
        int num_slices     = num_slices_all[i];
        const int gx       = CEIL_DIV(grid_size, num_slices);
        const int gy       = num_slices;
        max_num_block_sums = max(max_num_block_sums, gx * gy);
    }

    return max_num_block_sums;
}
template <class T>
__device__ inline float global_norm_squared_for_range(const T* data, size_t count) {
    size_t index      = blockIdx.x * blockDim.x + threadIdx.x;
    size_t grid_width = blockDim.x * gridDim.x;
    float accumulator = 0.f;
    for (size_t i = index; i < count; i += grid_width) {
        accumulator += (float)data[i] * (float)data[i];
    }
    // block-level reduce
    return blockReduce<warpReduceSum>(accumulator);
}
template <class T>
__global__ static void global_norm_squared_kernel(float* out, const T* data, size_t count, ptrdiff_t stride) {
    float block_sum = global_norm_squared_for_range(data + blockIdx.y * stride, count);
    // each block accumulates its partial sum to out[out_index]
    // we want to avoid using atomic add here so we combine this kernel with another kernel call
    // that sums up the partial block sums
    if (threadIdx.x == 0) {
        size_t out_index = blockIdx.y * gridDim.x + blockIdx.x;
        out[out_index]   = out[out_index] + block_sum;
    }
}
template <typename T>
inline float global_norm_squared(float* norm2, const T* values, size_t count, ptrdiff_t stride, int num_slices, int max_num_block_sums, bool reset,
                                 hipStream_t stream) {
    constexpr int block_size = 512;  // 256 may be better for shared memory of CU_x2_
    // launch just enough blocks to fill the grid. deliberately no DIV_CEIL.
    // having one block less than possible is a tiny performance hit, having
    // one block too many is catastrophic, since it only can start once all the other
    // blocks finish. anyway, I think cuda_threads_per_SM should be a multiple of 512
    // on all gpus, so the division really is going to be exact.
    auto now = GST_us();
    float a = 0, b = 0;
    if (1) {
        const int grid_size = deviceProp.maxThreadsPerMultiProcessor * deviceProp.multiProcessorCount / block_size;
        assert(grid_size > 0);  // gives a better error than letting the call below fail
        const int gx = CEIL_DIV(grid_size, num_slices), gy = num_slices;
        assert(gx * gy < 1024);  // we want to later accumulate the block sums in a single block
        if (reset) {
            cudaCheck(hipMemsetAsync(norm2, 0, max_num_block_sums * sizeof(float), stream));
        }
        global_norm_squared_kernel<<<dim3(gx, gy), block_size, 0, stream>>>(norm2, values, count, stride);
        cudaCheck(hipGetLastError());
        global_sum_deterministic(norm2, norm2, max_num_block_sums, main_stream);
        cudaCheck(hipMemcpy(&a, norm2, sizeof(float), hipMemcpyDeviceToHost));
    } else {
        size_t smemPB = 1024 * sizeof(float);
        cudaCheck(hipMemset(norm2, 0, sizeof(float)));
        CU_x2_<T, block_size><<<CEIL_DIV(count, block_size), block_size, smemPB, main_stream>>>(norm2, values, count);
        cudaCheck(hipMemcpy(&a, norm2, sizeof(float), hipMemcpyDeviceToHost));
        hipStreamSynchronize(main_stream);
    }
    // SUM::tX1 += GST_us()-now;
    return a;
}

double tNormOf(const std::vector<hGTensor>& tensors, int flag) {
    float *grad_norm_squared, a, a_pre = 0.0;
    grad_norm_squared = (float*)(GTensor::bt4c->data);
    double norm       = 0.0f;
    int num_slices[2] = {1, 1}, max_num_block_sums = get_max_num_block_sums(num_slices, 2);
    size_t nz          = 0;
    bool is_first_pass = true;  // i==0
    for (auto tensor : tensors) {
        assert(0);  // Deprecated
        /*//ShardInfo shard ={0, tensor->size()};
        size_t nEle = tensor->size();       nz+=nEle;
        assert(tensor->grad!=nullptr);
        floatX* val = (floatX*)(tensor->grad);
        global_norm_squared(grad_norm_squared, val, nEle, 0, 1,max_num_block_sums, is_first_pass, main_stream);
        if(DEBUG.check_tensor_norm){
            cudaCheck(hipMemcpy(&a, grad_norm_squared, sizeof(float), hipMemcpyDeviceToHost));
            assert(a>=a_pre);
            tensor->gnorm = sqrt(a-a_pre);           a_pre = a;
        }
        is_first_pass = false;*/
        // PrintTensor<floatX>("tNormOf",val,true,nEle,1);
        // break;
    }
    global_sum_deterministic(grad_norm_squared, grad_norm_squared, max_num_block_sums, main_stream);
    cudaCheck(hipMemcpy(&a, grad_norm_squared, sizeof(float), hipMemcpyDeviceToHost));

    norm = sqrt(a);
    a    = sqrt(a / nz);
    return norm;
}

//  TODO: Fuse to sgdv_update
double tNormOf(const hGTensor tensor, int flag) {
    float a, *norm2 = (float*)(GTensor::bt4c->data);
    int num_slices[2] = {1, 1}, max_num_block_sums = get_max_num_block_sums(num_slices, 2);
    size_t nz          = 0;
    bool is_first_pass = true;
    // ShardInfo shard ={0, tensor->size()};
    size_t nEle = tensor->size();
    nz += nEle;
    assert(tensor->grad != nullptr);
    // _INFO("|%s|=...",tensor->name);
    if (tensor->grad != nullptr) {
        int block_size = 1024, grid_size = deviceProp.maxThreadsPerMultiProcessor * deviceProp.multiProcessorCount / block_size;
        a = global_norm_squared(norm2, (floatX*)(tensor->grad), nEle, 0, 1, max_num_block_sums, is_first_pass, main_stream);
        // _INFO("\r|%s|=%g\n",tensor->name,a);
        tensor->gnorm = sqrt(a);
        a             = sqrt(a / nz);
    }
    if (tensor->data != nullptr) {
        a             = global_norm_squared(norm2, (floatX*)(tensor->data), nEle, 0, 1, max_num_block_sums, is_first_pass, main_stream);
        tensor->wnorm = sqrt(a);
    }

    return tensor->gnorm;
}

hGTensor huTensor::GetRow(hGTensor hOut, hGTensor token, hGTensor pos, int flag) { return hOut; }

void huTensor::Print(const string& title, int x, int flag, size_t nEle) const {
    bool isDevice = !isAtHost();
    switch (type) {
        case typNUMBER::F8E5M2:
            //    PrintTensor<__hip_fp8_e5m2_fnuz>(title.c_str(),(__hip_fp8_e5m2_fnuz *)data, isDevice,ne[0],ne[1],ne[2],ne[3],flag);
            PrintTensor<f8e5m2_t>(title.c_str(), (f8e5m2_t*)data, isDevice, ne[0], ne[1], ne[2], ne[3], flag);
            break;
        default:
            GTensor::Print(title, x, flag, nEle);
            break;
    }
}

huTensor::~huTensor() { Free(); }
template <class T, int NUM_THREADS = CU_T4B_SMALL>
__global__ static void CU_ternary_(float* gama, T* mat, int M, int N, int update) {  // block version
    int tid = threadIdx.x, warp = tid / WARP_SIZE, lane = tid % WARP_SIZE;
    // if(tid==0){//only for debug
    // 	for (int j = 0; j < M; j ++) {
    // 		gama[j] = 1.0;
    // 	}
    // 	return;
    // }
    int idx = blockIdx.x * NUM_THREADS + tid, ldJ = blockDim.x;
    T ta = (T)1.0, tb = (T)(-1.0), t0 = (T)(0.0);
    for (int j = tid; j < M; j += ldJ) {
        float sum = 0.0f, a, average = 0.0f;
        T* x0 = mat + j * N;
        for (int k = 0; k < N; k++) {
            a = CU_T2Float(x0 + k);
            sum += fabs(a);
        }
        average = (sum / (N)) + 1.0e-5;

        if (update == QUANT_ALG::W_SCALE) {
            gama[j] = average;
            ta = (T)(average), tb = (T)(-average);
        } else {
            gama[j] = 1.0f;
        }
        for (int k = 0; k < N; k++) {
            a     = CU_T2Float(x0 + k);
            x0[k] = a > average / 2 ? ta : a < -average / 2 ? tb : t0;
        }
    }
    __syncthreads();
}

template <class T, int NUM_THREADS = CU_T4B_SMALL>
__global__ static void CU_ternary_v0(float* out, T* x0, int N) {  // block version
    int tid = threadIdx.x, warp = tid / WARP_SIZE, lane = tid % WARP_SIZE;
    int idx = blockIdx.x * NUM_THREADS + tid, blockSize = blockDim.x;
    // if(idx >= N) { return; }

    __shared__ float average;
    // __shared__ T Ta,Tb;
    float sum = 0.0f, a;
    for (int j = tid; j < N; j += blockSize) {
        a = CU_T2Float(x0 + j);
        sum += fabs(a);  //	6.5
                         // sum += (float)(x0[j]);				//	6.3
    }
    float block_sum = blockReduce<warpReduceSum>(sum, true);
    // if (tid == 0) atomicAdd(out, block_sum);
    // SYNC_GRID();
    if (tid == 0) {
        *out    = block_sum;
        average = (*out / (N)) + 1.0e-5;
        out[1]  = average;
        //  average = average/2;
    }
    for (int j = tid; j < N; j += blockSize) {
        a     = CU_T2Float(x0 + j);
        x0[j] = a > average / 2 ? (T)1.0 : a < -average / 2 ? (T)(-1.0) : (T)(0.0);
    }
    __syncthreads();
}

template <class T, int NUM_THREADS = CU_T4B_SMALL>
__global__ static void CU_binary_(float* out, T* x0, size_t N) {  // block version
    int tid = threadIdx.x, warp = tid / WARP_SIZE, lane = tid % WARP_SIZE;
    int idx = blockIdx.x * NUM_THREADS + tid, blockSize = blockDim.x;
    // if(idx >= N) { return; }

    __shared__ float average;
    // __shared__ T Ta,Tb;
    float sum = 0.0f, a;
    for (int j = tid; j < N; j += blockSize) {
        a = CU_T2Float(x0 + j);  //	6.5
        sum += (float)(a);       //	6.3
    }
    float block_sum = blockReduce<warpReduceSum>(sum, true);
    // if (tid == 0) atomicAdd(out, block_sum);
    // SYNC_GRID();
    if (tid == 0) {
        *out    = block_sum;
        average = (*out / (N)) + 1.0e-5;
        out[1]  = average;
        //  average = average/2;
    }
    for (int j = tid; j < N; j += blockSize) {
        a     = CU_T2Float(x0 + j);
        x0[j] = a - average > 0 ? (T)1.0 : (T)(-1.0);
    }
    __syncthreads();
}

bool huTensor::ToTernary(int flag) {
    if (!BIT_TEST(flags, GTensor::F_TERNARY))
        return false;

    if (gama_T == nullptr)
        Alloc_1((void**)(&gama_T), false, sizeof(float) * ne[0]);

    assert(this->isParam() && gama_T != nullptr);
    assert(ne[2] == 1 && ne[3] == 1);  // only for 2D weight
    // Print("Before",0,-1);
    size_t count = size(), dBLOCK = CU_T4B_SMALL, smemPB = 1024 * sizeof(float);
    auto dGRID         = 1;  // hFish->curDevice()->GridDim(count);
    void* kernelArgs[] = {(void*)&gama_T, (void*)&data, (void*)&ne[0], (void*)&ne[1], (void*)&tpQuant};
    hipError_t err;
    switch (type) {
        case typNUMBER::T_SIGN:
            break;
        case typNUMBER::F16:
            assert(0);
            // CU_ternary_<<<, , 0, main_stream>>>(xxx,(__hip_bfloat16*)data, count);
            break;
        case typNUMBER::BF16:
            err = hipLaunchCooperativeKernel((void*)CU_ternary_<__hip_bfloat16>, dGRID, dBLOCK, kernelArgs, smemPB, main_stream);
            break;
        case typNUMBER::F8E5M2:
            err = hipLaunchCooperativeKernel((void*)CU_ternary_<__hip_fp8_e5m2_fnuz>, dGRID, dBLOCK, kernelArgs, smemPB, main_stream);
            // CU_ternary_<<<CEIL_DIV(count,CU_T4B_SMALL), CU_T4B_SMALL, 0, main_stream>>>(xxx,(__hip_fp8_e5m2_fnuz*)data, count);
            break;
        default:
            assert(0);
            break;
    }

    cudaCheck(err);
    D2H(gama_T, info, sizeof(info));
    if (tpQuant == QUANT_ALG::W_NOSCALE) {
        assert(info[0] == 1.0);
    }
    // Print(name,0,-1);
    // type = typNUMBER::T_SIGN;
    return true;
}