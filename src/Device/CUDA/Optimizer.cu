#include "hip/hip_runtime.h"
/**
 *  SPDX-FileCopyrightText: 2023-2025 Yingshi Chen <gsp.cys@gmail.com>
 *  SPDX-License-Identifier: MIT
 *
 *  Some idea is from https://github.com/karpathy/llm.c/blob/master/train_gpt2.cu
 *
 *  \brief cuda kernel of Optimizer
 *  \author Yingshi Chen
 */
#include "./kernel/Operator.cuh"
// #include "./llm_c/sampler.h"
#include "../../Manifold/Fish.hpp"
#include "../../Manifold/Neuron.hpp"
#include "../../Manifold/Optimizer.hpp"
#include "../Pipe.hpp"
#include "./kernel/utils.cuh"
extern unsigned long long rng_state;

typedef struct {
    ptrdiff_t offset;
    size_t size;
} ShardInfo;
//  reset grad online
template <typename Tp>
__device__ void sgd_update(Tp* params, float* tmp, Tp* grads0, size_t num_parameters, float learning_rate, float beta1, float beta2, float beta1_correction,
                           float beta2_correction, float eps, float weight_decay, float grad_scale, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_parameters) {
        return;
    }  // guard

    float grad      = grad_scale * (float)grads0[idx];
    float old_param = (tmp != NULL) ? tmp[idx] : (float)params[idx];
    float param     = old_param - (learning_rate * grad + weight_decay * old_param);
    // stochastic_rounding(param, &params[idx], seed);
    params[idx] = (Tp)(param);
    grads0[idx] = (Tp)(0.0);
    if (tmp != NULL) {
        tmp[idx] = param;
    }
}
//  reset grad online
template <typename Tp>
__global__ void CU_sgd(Tp* params, float* tmp, Tp* grads0, size_t num_parameters, ptrdiff_t w_stride, ptrdiff_t g_stride, ptrdiff_t s_stride,
                       float learning_rate, float beta1, float beta2, float beta1_correction, float beta2_correction, float eps, float weight_decay,
                       float grad_scale, unsigned int seed) {
    sgd_update(params + blockIdx.y * w_stride, tmp ? tmp + blockIdx.y * s_stride : NULL, grads0 + blockIdx.y * g_stride, num_parameters, learning_rate, beta1,
               beta2, beta1_correction, beta2_correction, eps, weight_decay, grad_scale, seed);
}

template <typename Tp, typename Tmv>
__global__ void CU_sgdv(Tp* params, Tp* grads0, Tmv* gv, size_t num_parameters, float learning_rate, float beta1, float beta2, float beta1_correction,
                        float beta2_correction, float eps, float weight_decay, float grad_scale, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_parameters) {
        return;
    }  // guard

    float grad = grad_scale * (float)grads0[idx];
    float v    = gv[idx];
    v          = lerp(grad * grad, v, beta2);  // beta2*v+(1-beta2)*grad*grad;
    gv[idx]    = v;
    v /= beta2_correction;  // v_hat
    float old_param = (float)params[idx];
    float param     = old_param - (learning_rate * (grad / (sqrtf(v) + eps) + weight_decay * old_param));
    // stochastic_rounding(param, &params[idx], seed);
    params[idx] = (Tp)(param);
    grads0[idx] = (Tp)(0.0);
}
template <typename Tp, typename Tmv>
__global__ void CU_lion_(Tp* params, Tp* grads0, Tmv* gm, size_t num_parameters, float learning_rate, float beta1, float beta2, float eps, float weight_decay,
                         float grad_scale, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_parameters) {
        return;
    }  // guard

    float grad = grad_scale * (float)grads0[idx];
    float m    = gm[idx], c;
    /*  Cautious LION
        mask = (update * grad > 0).to(grad.dtype)
        mask = mask * (mask.numel() / (mask.sum() + 1))
    */
    c = lerp(grad, m, beta1);  // beta1*m+(1-beta1)*grad;
    // c                 = c > 0 ? 1 : c == 0 ? 0 : -1;
    c               = c > eps ? 1 : c < -eps ? -1 : 0;  // ternary
    gm[idx]         = lerp(grad, m, beta2);             // beta2*m+(1-beta2)*grad;
    float old_param = CU_T2Float(params + idx);
    params[idx]     = CU_Float2T<Tp>(old_param - learning_rate * (c + weight_decay * old_param), seed);
    grads0[idx]     = (Tp)(0.0);
}

template <typename Tp, typename Tmv>
__global__ void CU_adamw_(Tp* params, float* tmp, Tp* grads0, Tmv* gm, Tmv* gv, size_t num_parameters, float learning_rate, uint64_t flags, float beta1,
                          float beta2, float beta1_correction, float beta2_correction, float eps, float weight_decay, float grad_scale, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_parameters) {
        return;
    }  // guard

    float grad = grad_scale * CU_T2Float(grads0 + idx), m = gm[idx], v = gv[idx];
    m = lerp(grad, m, beta1), gm[idx] = m;
    v = lerp(grad * grad, v, beta2), gv[idx] = v;
    m /= beta1_correction;  // m_hat
    v /= beta2_correction;  // v_hat
    float old_param = (tmp != NULL) ? tmp[idx] : (float)params[idx];
    float step = m / (sqrtf(v) + eps), T_spike = 50;
    //  Automatic detection of training instability
    // step = step>T_spike ?  T_spike : step<-T_spike ? -T_spike : step;
    float param = old_param - learning_rate * weight_decay * old_param - learning_rate * step;
    //  stochastic_rounding(param, &params[idx], seed);
    params[idx] = CU_Float2T<Tp>(param, seed);
    // params[idx] = (Tp)(param);
    grads0[idx] = (Tp)(0.0);

    if (tmp != NULL) {
        tmp[idx] = param;
    }
}

template <typename Tp, typename Tmv>
__device__ inline float _adamw_idx(float old_param, const PIPE_Optimizer<Tp, Tmv>& pipe, float& m, float& v, int idx) {
    m /= pipe.beta1_correction, v /= pipe.beta2_correction;  // m_hat    v_hat
    // float old_param = (float)pipe.params[idx];
    float step = m / (sqrtf(v) + pipe.eps);
    //  Automatic detection of training instability
    // step = step>T_spike ?  T_spike : step<-T_spike ? -T_spike : step;
    float param = old_param - pipe.learning_rate * pipe.weight_decay * old_param - pipe.learning_rate * step;
    //  stochastic_rounding(param, &params[idx], seed);
    param = CU_Float2T<Tp>(param, pipe.seed);
    // params[idx] = (Tp)(param);
    pipe.grads0[idx] = (Tp)(0.0);
    return param;
}

template <typename Tp, typename Tmv>
__global__ void CU_adamw_p(PIPE_Optimizer<Tp, Tmv> pipe) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= pipe.num_parameters) {
        return;
    }  // guard

    float grad = pipe.grad_scale * CU_T2Float(pipe.grads0 + idx), m = pipe.gm[idx], v = pipe.gv[idx];
    m = lerp(grad, m, pipe.beta1), pipe.gm[idx] = m;
    v = lerp(grad * grad, v, pipe.beta2), pipe.gv[idx] = v;
    // m /= pipe.beta1_correction;  // m_hat
    // v /= pipe.beta2_correction;  // v_hat
    float x = _adamw_idx((float)pipe.params[idx], pipe, m, v, idx), x2 = x * x;
    pipe.params[idx] = x;
    float block_sum  = blockReduce_v0<warpReduceSum>(x2, true);
    if (idx == 0)
        atomicAdd(pipe.wNorms, block_sum);
}

// row-major  slow versioin
template <typename Tp, typename Tmv>
__global__ void CU_adamw_ternary(PIPE_Optimizer<Tp, Tmv> pipe) {
    int M = pipe.ne[0], N = pipe.ne[1], tid = threadIdx.x;
    int idrow = blockIdx.x * blockDim.x + tid, offset = idrow * N;
    if (idrow >= M)
        return;  // guard
    float average = pipe.gama_T[idrow];
    Tp ta = (Tp)(average), tb = (Tp)(-average);
    char* terns  = (char*)(pipe.params) + offset / 8;
    Tp* params_x = pipe.paramX + offset;
    for (int k = 0; k < N; k += 8, offset += 8) {
        unsigned char tbyte = terns[k / 8];
#pragma unroll
        for (int kk = 0; kk < 8; kk++) {
            int bit         = BYTE_bit(tbyte, kk);  //(tbyte >> (7-kk)) & 0x1;
            float old_param = bit ? ta : tb;
            // CU_Float2T<Tp>(bit ? ta : tb, pipe.seed);      //
            int idx    = offset + kk;
            float grad = pipe.grad_scale * CU_T2Float(pipe.grads0 + idx), m = pipe.gm[idx], v = pipe.gv[idx];
            m = lerp(grad, m, pipe.beta1), pipe.gm[idx] = m;
            v = lerp(grad * grad, v, pipe.beta2), pipe.gv[idx] = v;
            // m /= pipe.beta1_correction, v /= pipe.beta2_correction;
            params_x[k + kk] = _adamw_idx(old_param, pipe, m, v, idx);
        }
    }
    CU_X2ternary_row(pipe.gama_T + idrow, params_x, terns, N);
    // __syncthreads();
}

template <typename Tp, typename Tmv>
__global__ void CU_adamw_s(PIPE_Optimizer<Tp, Tmv> pipe) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= pipe.num_parameters) {
        return;
    }  // guard

    float grad = pipe.grad_scale * CU_T2Float(pipe.grads0 + idx), m = pipe.gm[idx], v;
    v = lerp(grad * grad, m * m, pipe.beta2);
    m = lerp(grad, m, pipe.beta1), pipe.gm[idx] = m;
    // m /= pipe.beta1_correction;  // m_hat
    // v /= pipe.beta2_correction;  // v_hat
    float x = _adamw_idx((float)pipe.params[idx], pipe, m, v, idx), x2 = x * x;
    pipe.params[idx] = x;
    float block_sum  = blockReduce_v0<warpReduceSum>(x2, true);
    if (idx == 0)
        atomicAdd(pipe.wNorms, block_sum);
}

template <typename Tp, typename Tmv>
__global__ static void CU_adamw_Tile_v0(PIPE_Optimizer<Tp, Tmv> pipe) {
    const int TM = THREAD_TILE_M, TN = THREAD_TILE_N, thread_num = blockDim.x;
    int tid = threadIdx.x, idrow, idcol, M = pipe.ne[0], N = pipe.ne[1], trans = 1;
    idrow = blockIdx.x * TM + tid / TM;
    idcol = blockIdx.y * TN + tid % TM;
    if (idrow >= M || idcol >= N)
        return;  // guard
    fnPOS pA = trans == 0 ? fnCR2POS : fnRC2POS;
    int pos = pA(idrow, idcol, M, N), idx = pos, gpos = blockIdx.x * gridDim.y + blockIdx.y;
    float old_param = pipe.gama_T[gpos];
    float grad = pipe.grad_scale * CU_T2Float(pipe.grads0 + idx), m = pipe.gm[idx], v = pipe.gv[idx];
    m = lerp(grad, m, pipe.beta1), pipe.gm[idx] = m;
    v = lerp(grad * grad, v, pipe.beta2), pipe.gv[idx] = v;
    // m /= pipe.beta1_correction, v /= pipe.beta2_correction;
    float a   = _adamw_idx(old_param, pipe, m, v, idx);
    float sum = blockReduce_v0<warpReduceSum>(a, true);
    if (tid == 0) {
        pipe.gama_T[gpos] = sum / TM / TN;
    }
}

#define RC2TILE(r, c) (((r) / THREAD_TILE_M) * gridDim.y + ((c) / THREAD_TILE_N))
//  all element in tile has one mv
template <typename Tp, typename Tmv>
__global__ static void CU_adamw_Tile(PIPE_Optimizer<Tp, Tmv> pipe) {
    const int TM = THREAD_TILE_M, TN = THREAD_TILE_N, thread_num = blockDim.x;
    int tid = threadIdx.x, idrow, idcol, M = pipe.ne[0], N = pipe.ne[1], trans = 1;
    // const int nWrapT = std::min(WARP_SIZE,THREAD_TILE_M*THREAD_TILE_N);
    idrow = blockIdx.x * TM + tid / TM;
    idcol = blockIdx.y * TN + tid % TM;
    if (idrow >= M || idcol >= N)
        return;  // guard
    fnPOS pA = trans == 0 ? fnCR2POS : fnRC2POS;
    int pos = pA(idrow, idcol, M, N), idx = pos, gpos = RC2TILE(idrow, idcol);  // blockIdx.x * gridDim.y + blockIdx.y;
    float old_param = pipe.gama_T[gpos], m = pipe.gm[gpos], v = pipe.gv[gpos];
    float grad = pipe.grad_scale * CU_T2Float(pipe.grads0 + idx);
    float sum =
        CU_BlockSum<THREAD_TILE_M * THREAD_TILE_N>(grad);  // nWrapT<=WARP_SIZE ? warpReduceSum<nWrapT>(grad) : blockReduce_v0<warpReduceSum>(grad, true);
    grad = sum / TM / TN;
    m = lerp(grad, m, pipe.beta1), pipe.gm[gpos] = m;
    v = lerp(grad * grad, v, pipe.beta2), pipe.gv[gpos] = v;
    float a = _adamw_idx(old_param, pipe, m, v, idx);
    sum     = CU_BlockSum<THREAD_TILE_M * THREAD_TILE_N>(a);  // nWrapT<=WARP_SIZE ? warpReduceSum<nWrapT>(a) :blockReduce_v0<warpReduceSum>(a, true);
    if (tid == 0) {
        a                 = sum / TM / TN;
        pipe.gama_T[gpos] = a;  // CU_Float2T<Tp>(a, pipe.seed);   //
        atomicAdd(pipe.wNorms, a * a * TM * TN);
    }
}

//  all element in tile has one mv
template <typename Tp, typename Tmv>
__global__ static void CU_adamw_Tile_RC(PIPE_Optimizer<Tp, Tmv> pipe) {
    const int TM = THREAD_TILE_M, TN = THREAD_TILE_N, thread_num = blockDim.x;
    int tid = threadIdx.x, idrow, idcol, idrow_0, idcol_0, M = pipe.ne[0], N = pipe.ne[1], trans = 1;

    idrow = blockIdx.x * TM + tid / TM, idcol = blockIdx.y * TN + tid % TM;
    idrow_0 = blockIdx.x * TM + tid / TM + pipe.tile_r1;
    idcol_0 = blockIdx.y * TN + tid % TM + pipe.tile_c1;
    if (idrow >= M || idcol >= N)
        return;  // guard
    if (idrow_0 < 0)
        idrow_0 = 0;
    if (idrow_0 >= M)
        idrow_0 = M - 1;
    if (idcol_0 < 0)
        idcol_0 = 0;
    if (idcol_0 >= N)
        idcol_0 = N - 1;

    fnPOS pA   = trans == 0 ? fnCR2POS : fnRC2POS;
    int gpos_0 = RC2TILE(idrow_0, idcol_0);
    int pos_0 = pA(idrow_0, idcol_0, M, N), idx_0 = pos_0, gpos = RC2TILE(idrow, idcol);  // blockIdx.x * gridDim.y + blockIdx.y;
    if (gpos_0 != gpos) {
        int debug = 0;
    }
    float old_param = pipe.gama_T[gpos_0], m = pipe.gm[gpos_0], v = pipe.gv[gpos_0];
    float grad = pipe.grad_scale * CU_T2Float(pipe.grads0 + idx_0);
    float sum  = CU_BlockSum<THREAD_TILE_M * THREAD_TILE_N>(grad);  // blockReduce_v0<warpReduceSum>(grad, true);
    grad       = sum / TM / TN;
    m = lerp(grad, m, pipe.beta1), v = lerp(grad * grad, v, pipe.beta2);
    float sum_m = CU_BlockSum<THREAD_TILE_M * THREAD_TILE_N>(m);  // blockReduce_v0<warpReduceSum>(m, true);
    float sum_v = CU_BlockSum<THREAD_TILE_M * THREAD_TILE_N>(v);  // blockReduce_v0<warpReduceSum>(v, true);
    float a     = _adamw_idx(old_param, pipe, m, v, idx_0);
    sum         = CU_BlockSum<THREAD_TILE_M * THREAD_TILE_N>(a);  // blockReduce_v0<warpReduceSum>(a, true);
    if (tid == 0) {
        assert(!(isnan(sum) || isinf(sum)));
        assert(!(isnan(sum_v) || isinf(sum_v)));
        assert(!(isnan(sum_m) || isinf(sum_m)));
        pipe.gama_T[gpos] = sum / TM / TN;
        pipe.gv[gpos]     = sum_v / TM / TN;
        pipe.gm[gpos]     = sum_m / TM / TN;
    }
}
//  each element in tile has different mv
template <typename Tp, typename Tmv>
__global__ static void CU_adamw_Tile_each_mv(PIPE_Optimizer<Tp, Tmv> pipe) {
    const int TM = THREAD_TILE_M, TN = THREAD_TILE_N, thread_num = blockDim.x;
    int tid = threadIdx.x, idrow, idcol, M = pipe.ne[0], N = pipe.ne[1], trans = 1;
    idrow = blockIdx.x * TM + tid / TM;
    idcol = blockIdx.y * TN + tid % TM;
    if (idrow >= M || idcol >= N)
        return;  // guard
    fnPOS pA = trans == 0 ? fnCR2POS : fnRC2POS;
    int pos = pA(idrow, idcol, M, N), idx = pos, gpos = blockIdx.x * gridDim.y + blockIdx.y;
    float old_param = pipe.gama_T[gpos], m = pipe.gm[gpos], v = pipe.gv[gpos];
    float grad = pipe.grad_scale * CU_T2Float(pipe.grads0 + idx);
    m = lerp(grad, m, pipe.beta1), v = lerp(grad * grad, v, pipe.beta2);

    float sum   = blockReduce_v0<warpReduceSum>(grad, true);
    float sum_m = blockReduce_v0<warpReduceSum>(m, true);
    float sum_v = blockReduce_v0<warpReduceSum>(v, true);
    float a     = _adamw_idx(old_param, pipe, m, v, idx);  //  m,v => m_hat,v_hat
    sum         = blockReduce_v0<warpReduceSum>(a, true);
    if (tid == 0) {
        pipe.gama_T[gpos] = sum / TM / TN;
        pipe.gv[gpos]     = sum_v / TM / TN;
        pipe.gm[gpos]     = sum_m / TM / TN;
    }
}

// bool Fuyou::Exploitation(hGensor cur, int flag) {
//     int nP = cur->size(), dT4B = 512 ,nF = cur->fuyous.size();  //
//     int dGRID = CEIL_DIV(nP, dT4B);
//     for (auto t : cur->fuyous) {
//         //  position[i] = alpha*A->position[i] + beta*B->position[i];
//         CU_mix_<<<dGRID, dT4B, 0, main_stream>>>(alpha, ToX(cur), beta, ToX(t),nP);
//     }
//     return true;
// }

bool Fuyou::Exploitation(hGensor tHead, hGensor tNext, int flag) {
    if (!tHead->is2D())
        return false;
    int nParam = tHead->size(), dT4B = 512, M = tHead->ne[0], N = tHead->ne[1], nRander = M;  //
    // int dGRID = CEIL_DIV(nParam, dT4B);
    int mGRID = CEIL_DIV(M, dT4B),pGRID = CEIL_DIV(nParam, dT4B);

    hiprandState* d_states;
    cudaCheck(hipMalloc(&d_states, nRander * sizeof(hiprandState)));
    seed = rander.RandU32();
    CU_initrand<<<CEIL_DIV(nRander, 256), 256>>>(d_states, seed, nRander);

    switch (params.algorithm) {
        case Fuyou_params::GENE_MIX:
            CU_mix_<<<pGRID, dT4B, 0, main_stream>>>(params.alpha, ToX(tNext), 1.0 - params.alpha, ToX(tHead), nParam);
            break;
        case Fuyou_params::GENE_MUTATION:
            // CU_mutation_<<<mGRID, dT4B, 0, main_stream>>>(d_states, T_mutation, ToX(tNext), nParam, N);
            break;
        case Fuyou_params::PARTICLE_GENETIC:
            CU_PSO_2D<<<mGRID, dT4B, 0, main_stream>>>(d_states, params.alpha, ToX(tNext), params.social, ToX(tHead), nParam, N);
            // CU_mutation_<<<mGRID, dT4B, 0, main_stream>>>(d_states, T_mutation, ToX(tNext), ToX(tHead), nParam, N);
            // why T_crossover=0.6 is still effective
            CU_crossover_<<<mGRID, dT4B, 0, main_stream>>>(d_states, params.T_crossover, ToX(tNext), ToX(tHead), nParam, N);    
            break;
        case Fuyou_params::PARTICLE_SWARM:
        default:
            // CU_mix_<<<dGRID, dT4B, 0, main_stream>>>(alpha, ToX(tNext), beta, ToX(tHead), nP);
            CU_PSO_2D<<<mGRID, dT4B, 0, main_stream>>>(d_states, params.alpha, ToX(tNext), params.social, ToX(tHead), nParam, N);
            break;
    }
    cudaCheck(hipFree(d_states));

    return true;
}

template <typename Tp, typename Tmv>
void Optimizer_update(PIPE_Optimizer<Tp, Tmv>& pipe, hipStream_t stream) {
    // hipError_t err       = hipSuccess;
    int64_t ne[4]         = {pipe.ne[0], pipe.ne[1], pipe.ne[2], pipe.ne[3]};
    int dT4B              = 512;  //  1024?
    int dGRID             = CEIL_DIV(pipe.num_parameters, dT4B);
    size_t smemPB         = 1024 * sizeof(float);
    pipe.beta1_correction = 1.0f - powf(pipe.beta1, pipe.iter);
    pipe.beta2_correction = 1.0f - powf(pipe.beta2, pipe.iter);

    D20(pipe.wNorms, sizeof(float) * 1);
    if (pipe.gm == nullptr) {  // SGD,SGD_V
        // if (gv == nullptr) {
        //     CU_sgd<<<num_blocks, block_size, 0, stream>>>(params, tmp, grads0, num_parameters, w_stride, g_stride,
        //                                                                     s_stride, learning_rate, beta1, beta2, beta1_correction, beta2_correction, eps,
        //                                                                     weight_decay, grad_scale, seed);
        // } else {
        //     CU_sgdv<<<num_blocks, block_size, 0, stream>>>(params, grads0, gv, num_parameters, learning_rate, beta1, beta2, beta1_correction,
        //                                                    beta2_correction, eps, weight_decay, grad_scale, seed);
        // }
    } else {  //   ADAM_S LION(locked!!!)
        if (pipe.gv == nullptr) {
            CU_adamw_s<<<dGRID, dT4B, 0, stream>>>(pipe);
            //  pipe.eps = pipe.grad_norm / pipe.num_parameters;    for lion
            // CU_lion_<<<num_blocks, block_size, 0, stream>>>(params, grads0, gm, num_parameters, learning_rate, beta1, beta2, eps, weight_decay,grad_scale,
            // seed);
        } else {
            if (pipe.isBitParam) {
                // PrintTensor<Tp>("grad", (Tp*)pipe.grads0, true, pipe.num_parameters, 1,1,1,-1);
                switch (pipe.tensor->type) {
                    case typNUMBER::T_BINARY_TILE: {
                        dim3 dBlock(THREAD_TILE_M * THREAD_TILE_N), dGrid(CEIL_DIV(ne[0], THREAD_TILE_M), CEIL_DIV(ne[1], THREAD_TILE_N));
                        CU_adamw_Tile<<<dGrid, dBlock, smemPB, stream>>>(pipe);
                    } break;
                    default:
                        if (DEBUG.T_ternary == 1) {
                            CU_adamw_p<<<dGRID, dT4B, 0, stream>>>(pipe);
                            CU_ternary_online<<<CEIL_DIV(pipe.ne[0], dT4B), dT4B, smemPB, stream>>>(pipe.params, pipe.ne[0], pipe.ne[1]);
                            // PrintTensor<floatX>(pipe.tensor->name, (floatX*)pipe.params, true, pipe.ne[0], pipe.ne[1], pipe.ne[2], pipe.ne[3], -1);
                        } else {
                            CU_adamw_ternary<<<CEIL_DIV(ne[0], dT4B), dT4B, smemPB, stream>>>(pipe);
                            // pipe.tensor->GetDataX(dump_flag,"w1");
                        }
                        break;
                }
            } else {  //  ADAMw
                //  void* kernelArgs[]    = {(void*)&pipe};
                // err = hipLaunchCooperativeKernel((void*)CU_adamw_<Tp,Tmv>, dGRID, dT4B, kernelArgs, smemPB, main_stream);
                // cudaCheck(err);      "too many blocks in cooperative launch"
                CU_adamw_p<<<dGRID, dT4B, 0, stream>>>(pipe);
            }
            D2e(pipe.wNorms, pipe.tensor->wnorm, 0x0);
            pipe.tensor->wnorm = sqrt(pipe.tensor->wnorm);
            // {            //  deparecated version result=/home/cys/rnd/lic/log/gpt2/0703_adamw.info
            //     CU_adamw_<<<dGRID, dT4B, 0, stream>>>(pipe.params, pipe.tmp, pipe.grads0, pipe.gm, pipe.gv, pipe.num_parameters,
            //     pipe.learning_rate,pipe.flags,
            //                                                  pipe.beta1, pipe.beta2, pipe.beta1_correction, pipe.beta2_correction, pipe.eps,
            //                                                  pipe.weight_decay, pipe.grad_scale, pipe.seed);
            // }
        }
    }
    cudaCheck(hipGetLastError());
}

void Optimizer::ClearOnCUDA(int flag) {}
void Optimizer::InitOnCUDA(int flag) {
    ADAM_params_ adam = TrainParams().adam;
    // GD_METHOD tpCurGD = tpGD;

    int C      = _fish->config.nEmbed();  // num_slices = 1,
    size_t off = 0;
    for (auto tensor : opt_ps) {
        size_t nP = tensor->size();  //, grid_size = CEIL_DIV(nP, 512);
        auto& im  = _fish->GetGensorInfo(tensor);
        if (tpGD == SGD_HYBRID) {
            // tpCurGD = im.isAdam ? ADAMw : SGD;
        }
        // if(tpCurGD==ADAMw){
        //     // _INFO("Optimizer allocating %zu MiB for m\n", (adam.n_parameters * sizeof(float)) >> 20);
        //     cudaCheck(hipMalloc((void**)&(im.gm), tensor->size() * sizeof(float)));
        //     cudaCheck(hipMemset(im.gm, 0, tensor->size() * sizeof(float)));
        // }
        // if(master_weights!=nullptr){
        //     copy_and_cast_kernel<<<dim3(grid_size, num_slices), 512, 0, main_stream>>>(master_weights+off, ToX(tensor), nP,nP, nP);
        //     cudaCheck(hipGetLastError());
        // }
        off += nP;
    }
}

//  Deprecated
int UpdateTensorParam_cuda(hGTensor tensor, Optimizer* hOPT, float& grad_norm, int flag) {
    CLI_params config   = hOPT->_fish->config;
    ADAM_params_ adam   = hOPT->TrainParams().adam;
    auto& im            = hOPT->_fish->GetGensorInfo(tensor);
    float learning_rate = hOPT->LearningRate(), beta1 = adam.beta1, beta2 = adam.beta2, eps = adam.eps;
    int iter          = hOPT->GetITER();              // num_slices = 1,
    unsigned int seed = hOPT->rRounding.RandInt32();  // random_u32(&rng_state);
    const char* name  = tensor->name;
    ShardInfo shard   = {0, tensor->size()};
    float wd          = adam.decay;  // we only want to weight decay the 2D tensors and leave all 1D tensors alone
    if (tensor->shape.size() == 1)
        wd = 0;

    floatX *param_ptr = ToX(tensor), *grad_ptr = ToG(tensor);
    ptrdiff_t opt_state_offset = tensor->offset;                           // multi_gpu_config->zero_stage < 1 ?  local_offset_full : local_offset_partial;
    floatMV *m_ptr = (floatMV*)tensor->gm, *v_ptr = (floatMV*)tensor->gv;  // gv==nullptr? nullptr : gv + opt_state_offset;
    // float* master_ptr = NULL;                                              // why this would slow down converge?
    // if (master_weights != NULL && im.isAdam) {
    //     master_ptr = master_weights + opt_state_offset;
    // }

    if (adam.clip_alg != 0 || config.lars_ratio > 0) {
        grad_norm     = tensor->Length(1);  //    tNormOf(tensor, 0x0);
        tensor->gnorm = grad_norm;
        if (fabs(grad_norm) < 1.0e-10) {
            _INFO("\t|g|=0@%s!", tensor->name);
        }
        if (isnan(grad_norm)) {
            _INFO("!!! NAN |g|@%s !!!\n", tensor->name);
        }
        if (grad_norm > adam.gclip) {
            // _INFO("\tdelta|%s|=%g scale=%g\n",tensor->name,grad_norm,adam.gclip/grad_norm);
        }
    }
    float grad_scale = (grad_norm > adam.gclip) ? adam.gclip / grad_norm : 1.0f;
    if (config.lars_ratio > 0) {
        grad_scale = tensor->rLARS(grad_scale, config.lars_ratio, 0x0);
    }
    PIPE_Optimizer<floatX, floatMV> pipe(shard.size, shard.size, shard.size, shard.size, tensor->flags, learning_rate, beta1, beta2, iter, eps, wd, grad_scale,
                                         grad_norm, seed);
    pipe.Update(tensor.get());
    Optimizer_update(pipe, main_stream);

    cudaCheck(hipGetLastError());
    return 0x0;
}

/*Deparecated
int RAW_update(std::vector<hGTensor>& tensors, Optimizer* hOPT, float& grad_norm, int alg, int flag) {
    CLI_params config = hOPT->_fish->config;
    ADAM_params_ adam = hOPT->TrainParams().adam;
    if (adam.clip_alg == 0)
        grad_norm = flag == 0x10002 ? 1.0e6 : tNormOf(tensors, 0x0);
    double gnorm_0 = grad_norm, gnorm_1 = 0;
    float learning_rate = hOPT->LearningRate();
    float beta1 = adam.beta1, beta2 = adam.beta2, eps = adam.eps, weight_decay = adam.decay * adam.alpha;
    NVTX_RANGE_FN();
    size_t np      = 0;
    int num_slices = 1, iter = hOPT->GetITER();

    // for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {        // generate a unique seed for each tensor
    for (auto tensor : tensors) {
        if (alg == 0) {
            UpdateTensorParam_cuda(tensor, hOPT, grad_norm, flag);
        } else {
            unsigned int seed = 42;  // random_u32(&rng_state);
            const char* name  = tensor->name;
            ShardInfo shard   = {0, tensor->size()};
            float wd          = weight_decay;  // we only want to weight decay the 2D tensors and leave all 1D tensors alone
            if (tensor->shape.size() == 1)
                wd = 0;
            // ptrdiff_t local_offset_full=0,local_offset_partial=tensor->offset;
            floatX* param_ptr = ToX(tensor);  //(floatX*)params + local_offset_full;
            floatX* grad_ptr  = ToG(tensor);  //(floatX*)grads0 + local_offset_full;

            ptrdiff_t opt_state_offset = np;  // multi_gpu_config->zero_stage < 1 ?  local_offset_full : local_offset_partial;

            // float* m_ptr = gm + opt_state_offset,* v_ptr = gv + opt_state_offset;
            float *m_ptr = (float*)tensor->gm, *v_ptr = (float*)tensor->gv;
            float* master_ptr = NULL;
            // if (master_weights != NULL) { master_ptr = master_weights + opt_state_offset; }

            if (adam.clip_alg != 0 || config.lars_ratio > 0) {
                grad_norm = tNormOf(tensor, 0x0);
                gnorm_1 += grad_norm * grad_norm;
            }
            float grad_scale = (grad_norm > adam.gclip) ? adam.gclip / grad_norm : 1.0f;
            // if( config.lars_ratio>0 && tensor->shape.size()>1){
            //     grad_scale = tensor->rLARS(config.lars_ratio,0x0);
            // }

            if (flag != 0x10001) {  // some debug
                Optimizer_update(param_ptr, master_ptr, grad_ptr, m_ptr, v_ptr, shard.size, shard.size, shard.size, shard.size,
                                 num_slices,  // num_parameters,ptrdiff_t w_stride, ptrdiff_t g_stride, ptrdiff_t s_stride,  int num_slices,
                                 learning_rate, beta1, beta2, iter, eps, wd, grad_scale, seed, main_stream);
            }
            cudaCheck(hipGetLastError());
        }
        np += tensor->size();
    }
    // assert(fabs(gnorm_1-gnorm_0*gnorm_0)<1.0e-6*gnorm_1);       // verify
    assert(np == adam.n_parameters);
    cudaCheck(hipDeviceSynchronize());
    return 0x0;
}
    */