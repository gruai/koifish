#include "hip/hip_runtime.h"
#include "./cuda_common.h"
#include "./cublas_common.h"
#include "./llm_c/matmul.cuh"
#include "./llm_c/layernorm.cuh"
#include "./llm_c/encoder.cuh"
#include "./llm_c/fused_classifier.cuh"
#include "./kernel/rope.cuh"
#include "../../Manifold/Neuron.hpp"
#include "../../Manifold/Fish.hpp"
#include "./EDevice.hpp"
#define NOMINMAX

// #undef ENABLE_CUDNN
#ifdef ENABLE_CUDNN
    #include "cudnn_frontend.h"
    #include "./llm_c/cudnn_att.h"
    namespace fe = cudnn_frontend;
    #if defined(ENABLE_FP32)
        // static_assert(false, "cuDNN is not supported in FP32 mode.")
        // use fp16 (note: this may require gradient scaler, currently not implemented!)
    #elif defined(ENABLE_FP16)
        #define CUDNN_16BIT fe::DataType_t::HALF
    #else // Default to bfloat16
        #define CUDNN_16BIT fe::DataType_t::BFLOAT16    
    #endif
#else
    // defines: attention_forward, attention_backward
    #include "./llm_c/attention.cuh"
#endif

static hipEvent_t cuStart, cuEnd;
#define fuMM matmul_forward_cublaslt

#ifdef ENABLE_CUDNN
static hipdnnHandle_t cudnn_handle;
static size_t cudnn_workspace_size = 0; // dynamically allocated as needed (up to 256MiB!)
static void* cudnn_workspace = NULL;

static void cuDNNCheck(hipdnnStatus_t error, const char *file, int line) {
    if (error != HIPDNN_STATUS_SUCCESS) {
        printf("[CUDNN ERROR] at file %s:%d:\n%s\n", file, line, hipdnnGetErrorString(error));
        exit(EXIT_FAILURE);
    }
};
#define cuDNNCheck(err) (cuDNNCheck(err, __FILE__, __LINE__))

static void checkCudnnFE(const fe::error_object& e, const char *file, int line) {
    if(!e.is_good()) {
        printf("[CUDNN ERROR] at file %s:%d:\n%s\n", file, line, e.err_msg.c_str());
        exit(EXIT_FAILURE);
    }
}
#define checkCudnnFE(err) checkCudnnFE(err, __FILE__, __LINE__)

enum UIDs {
    Q_UID,
    K_UID,
    V_UID,
    Attn_scale_UID,
    O_UID,
    Stats_UID,
    dO_UID,
    dQ_UID,
    dK_UID,
    dV_UID
};

// Need a cache because graph->build_operation_graph() is slow but everything else seems fast
using cache_type_fwd = std::map<std::tuple<int,int,int,int, int>, std::shared_ptr<fe::graph::Graph>>;
using cache_type_bwd = std::map<std::tuple<int,int,int,int>, std::shared_ptr<fe::graph::Graph>>;

// Loosely based on cuDNN frontend samples functions and massively simplified
auto lookup_cache_or_build_graph_fwd(int B,int H,int T,int HS, int is_inference_only) {

    static cache_type_fwd user_maintained_cache_fwd;

    auto key = std::make_tuple(B, H, T, HS, is_inference_only);

    auto it = user_maintained_cache_fwd.find(key);
    if (it != user_maintained_cache_fwd.end()) {
        return it->second;
    }

    auto graph = std::make_shared<fe::graph::Graph>();
#if defined(ENABLE_BF16)
    graph->set_io_data_type(CUDNN_16BIT)
          .set_intermediate_data_type(fe::DataType_t::FLOAT)
          .set_compute_data_type(fe::DataType_t::FLOAT);
#else
    assert(0);
#endif

    // QKV is (B, T, 3, NH, HS) which cuDNN can handle directly without an external permute
    auto Q = graph->tensor(fe::graph::Tensor_attributes().set_name("Q")
                               .set_dim({B, H, T, HS})
                               .set_uid(Q_UID)
                               .set_stride({3 * H * HS * T,  HS, 3 * H * HS, 1}));
    auto K = graph->tensor(fe::graph::Tensor_attributes().set_name("K")
                               .set_dim({B, H, T, HS})
                               .set_uid(K_UID)
                               .set_stride({3 * H * HS * T, HS, 3 * H * HS, 1}));
    auto V = graph->tensor(fe::graph::Tensor_attributes().set_name("V")
                               .set_dim({B, H, T, HS})
                               .set_uid(V_UID)
                               .set_stride({3 * H * HS * T, HS, 3 * H * HS, 1}));
    auto attn_scale = graph->tensor(fe::graph::Tensor_attributes().set_name("attn_scale")
                               .set_dim({1, 1, 1, 1})
                               .set_stride({1, 1, 1, 1})
                               .set_uid(Attn_scale_UID)
                               .set_is_pass_by_value(true)
                               .set_data_type(fe::DataType_t::FLOAT));

    auto sdpa_options = fe::graph::SDPA_attributes().set_name("flash_attention");
    sdpa_options.set_is_inference(is_inference_only);
    sdpa_options.set_attn_scale(attn_scale);
    sdpa_options.set_causal_mask(true);

    // Create the graph operation and get the output tensors back
    auto [O, stats] = graph->sdpa(Q, K, V, sdpa_options);

    // Output is (B, T, NH, HS) BF16/FP16 and stats for backward pass is (B, NH, T) FP32
    O->set_output(true).set_dim({B, H, T, HS}).set_stride({H * HS * T, HS, H * HS, 1}).set_uid(O_UID);

    assert(stats == nullptr || is_inference_only == false);
    if (is_inference_only == false) {
        stats->set_output(true).set_data_type(fe::DataType_t::FLOAT)
                               .set_dim({B, H, T, 1})
                               .set_stride({H * T, T, 1, 1})
                               .set_uid(Stats_UID);
    }

    checkCudnnFE(graph->validate());

    // Build the operation graph and execution part (this is the VERY SLOW PART)
    checkCudnnFE(graph->build_operation_graph(cudnn_handle));
    auto plans = graph->create_execution_plans({fe::HeurMode_t::A});
    checkCudnnFE(graph->check_support(cudnn_handle));
    checkCudnnFE(graph->build_plans(cudnn_handle));
    // Reallocate the workspace if the required size is greater than the current workspace
    // In H100 this may be around 16B
    if (graph->get_workspace_size() > cudnn_workspace_size) {
        if (cudnn_workspace_size > 0) {
            cudaCheck(hipFree(cudnn_workspace));
        }
        cudnn_workspace_size = graph->get_workspace_size();
        cudaCheck(hipMalloc(&cudnn_workspace, cudnn_workspace_size));
    }

    user_maintained_cache_fwd.insert({key, graph});

    return graph;
}

auto lookup_cache_or_build_graph_bwd(int B, int NH, int T, int HS) {
    static cache_type_bwd user_maintained_cache_bwd;

    auto key = std::make_tuple(B, NH, T, HS);

    auto it = user_maintained_cache_bwd.find(key);
    if (it != user_maintained_cache_bwd.end()) {
        return it->second;
    }

    auto graph = std::make_shared<fe::graph::Graph>();
#if defined(ENABLE_BF16)
    graph->set_io_data_type(CUDNN_16BIT)
          .set_intermediate_data_type(fe::DataType_t::FLOAT)
          .set_compute_data_type(fe::DataType_t::FLOAT);
#else
    assert(0);
#endif
    // (B, N, 3, NH, HS)
    // must come from inp (which means we also need to convert THAT to FP16)
    auto Q = graph->tensor(fe::graph::Tensor_attributes().set_name("Q")
                            .set_dim({B, NH, T, HS})
                            .set_uid(Q_UID)
                            .set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}));
    auto K = graph->tensor(fe::graph::Tensor_attributes().set_name("K")
                            .set_dim({B, NH, T, HS})
                            .set_uid(K_UID)
                            .set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}));
    auto V = graph->tensor(fe::graph::Tensor_attributes().set_name("V")
                            .set_dim({B, NH, T, HS})
                            .set_uid(V_UID)
                            .set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}));
    auto O = graph->tensor(fe::graph::Tensor_attributes().set_name("O")
                            .set_dim({B, NH, T, HS})
                            .set_uid(O_UID)
                            .set_stride({NH * HS * T, HS, NH * HS, 1}));
    auto dO = graph->tensor(fe::graph::Tensor_attributes().set_name("dO")
                            .set_dim({B, NH, T, HS})
                            .set_uid(dO_UID)
                            .set_stride({NH * HS * T, HS, NH * HS, 1}));

    auto stats = graph->tensor(fe::graph::Tensor_attributes().set_name("stats")
                            .set_dim({B, NH, T, 1})
                            .set_uid(Stats_UID)
                            .set_stride({NH * T, T, 1, 1})
                            .set_data_type(fe::DataType_t::FLOAT));
    auto attn_scale = graph->tensor(fe::graph::Tensor_attributes().set_name("attn_scale")
                            .set_dim({1, 1, 1, 1})
                            .set_stride({1, 1, 1, 1})
                            .set_is_pass_by_value(true)
                            .set_uid(Attn_scale_UID)
                            .set_data_type(fe::DataType_t::FLOAT));
    auto sdpa_backward_options = fe::graph::SDPA_backward_attributes().set_name("flash_attention_backward")
#if CUDNN_FRONTEND_MAJOR_VERSION > 1 || CUDNN_FRONTEND_MINOR_VERSION >= 5
                            .set_deterministic_algorithm(true) // 1.5+ needs this for determinism
#endif
                            .set_causal_mask(true)
                            .set_attn_scale(attn_scale);

    // Create the graph operation and get the output tensors back
    auto [dQ, dK, dV] = graph->sdpa_backward(Q, K, V, O, dO, stats, sdpa_backward_options);

    dQ->set_output(true).set_dim({B, NH, T, HS}).set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}).set_uid(dQ_UID);
    dK->set_output(true).set_dim({B, NH, T, HS}).set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}).set_uid(dK_UID);
    dV->set_output(true).set_dim({B, NH, T, HS}).set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}).set_uid(dV_UID);

    checkCudnnFE(graph->validate());

    // Build the operation graph and execution part (this is the VERY SLOW PART)
    checkCudnnFE(graph->build_operation_graph(cudnn_handle));
    auto plans = graph->create_execution_plans({fe::HeurMode_t::A});
    checkCudnnFE(graph->check_support(cudnn_handle));
    checkCudnnFE(graph->build_plans(cudnn_handle));

    // Reallocate the workspace if the required size is greater than the current workspace
    // By default, cuDNN uses up to 256MiB of workspace, so we don't want to just allocate the maximum
    if (graph->get_workspace_size() > cudnn_workspace_size) {
        if (cudnn_workspace_size > 0) {
            cudaCheck(hipFree(cudnn_workspace));
        }
        cudnn_workspace_size = graph->get_workspace_size();     //1008599040
        cudaCheck(hipMalloc(&cudnn_workspace, cudnn_workspace_size));
    }

    user_maintained_cache_bwd.insert({key, graph});
    return graph;
}

void attention_forward_cudnn(floatX* out,  // output: (B, T, NH, HS)
                             float* stats, // output for backward pass: (B, NH, T)
                             floatX* inp,  // input: (B, T, 3, NH, HS) QKV
                             int B, int T, int NH, int C, hipStream_t stream) {
    NVTX_RANGE_FN();
    int HS = C / NH; // number of features per head
    bool is_inference_only = (stats == nullptr);

    cuDNNCheck(hipdnnSetStream(cudnn_handle, stream));

    // Get graph and tensors from cache (or generate it on first use)
    auto graph = lookup_cache_or_build_graph_fwd(B, NH, T, HS, is_inference_only);

    // Prepare all the tensor pointers for executing the graph
    void* devPtrQ = inp;
    void* devPtrK = (inp + C);
    void* devPtrV = (inp + 2 * C);
    float attn_scale_cpu = 1.0 / sqrtf(HS);
    void* devPtrO = out;

    // Build variant pack
    std::unordered_map<int64_t , void*> variant_pack = {
        {Q_UID, devPtrQ}, {K_UID, devPtrK}, {V_UID, devPtrV}, {Attn_scale_UID, &attn_scale_cpu}, {O_UID, devPtrO}};

    // Add the stats tensor unless we are only doing inference (only needed for backward pass)
    if (is_inference_only == false) {
        variant_pack[Stats_UID] = stats;
    }

    // Execute graph
    checkCudnnFE(graph->execute(cudnn_handle, variant_pack, cudnn_workspace));
    cudaCheck(hipGetLastError());
}

void attention_backward_cudnn(floatX* dqkvr,                                       // output
                              floatX* dout, floatX* qkvr, floatX* o, float* stats, // inputs
                              int B, int T, int NH, int C, hipStream_t stream) {
    NVTX_RANGE_FN();
    int HS = C / NH; // number of features per head

    // Get graph and tensors from cache (or generate it on first use)
    auto graph = lookup_cache_or_build_graph_bwd(B, NH, T, HS);

    // Prepare all the tensor pointers for executing the graph
    void* devPtrQ = qkvr;
    void* devPtrK = (qkvr + NH * HS);
    void* devPtrV = (qkvr + 2 * NH * HS);
    void* devPtrO = o;
    void* devPtrdO = dout;
    void* devPtrStats = stats;
    float attn_scale_cpu = 1.0 / sqrtf(HS);

    void* devPtrdQ = dqkvr;
    void* devPtrdK = (dqkvr + NH * HS);
    void* devPtrdV = (dqkvr + 2 * NH * HS);

    // Build variant pack that links each tensor to its data pointer
    std::unordered_map<int64_t, void*> variant_pack = {
        {Q_UID, devPtrQ}, {K_UID, devPtrK}, {V_UID, devPtrV}, {O_UID, devPtrO}, {dO_UID, devPtrdO}, {Stats_UID, devPtrStats},
        {dQ_UID, devPtrdQ}, {dK_UID, devPtrdK}, {dV_UID, devPtrdV},
        {Attn_scale_UID, &attn_scale_cpu}};

    // Execute graph
    cuDNNCheck(hipdnnSetStream(cudnn_handle, stream));
    checkCudnnFE(graph->execute(cudnn_handle, variant_pack, cudnn_workspace));
    cudaCheck(hipGetLastError());
}

void create_cudnn() {
    cuDNNCheck(hipdnnCreate(&cudnn_handle));
}

void destroy_cudnn() {
    if (cudnn_workspace != NULL) { cudaCheck(hipFree(cudnn_workspace)); }
    cuDNNCheck(hipdnnDestroy(cudnn_handle));
}
#endif


bool InitCUDA(const CLI_params&hparams,EDGE_DEVICES *hDevice,int flag){
    //  hipDriverGetVersion
    //  hipRuntimeGetVersion
    int local_device_idx = 0, override_enable_tf32 = 1;
    hipError_t err = hipSetDevice(0);
    if (err != hipSuccess) {
        printf("[InitCUDA] failed at hipSetDevice! ERR=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    cudaCheck(hipGetDeviceProperties(&deviceProp, local_device_idx));
    if (1) {
        printf("[System]\n");
        printf("Device %d: %s\n", local_device_idx, deviceProp.name);
    }

    // set up the cuda streams. atm everything is on the single main stream
    cudaCheck(hipStreamCreate(&main_stream));
    nvtxNameCudaStreamA(main_stream, "main stream");

    // set up cuBLAS and cuBLASLt
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    bool enable_tf32 = FLOAT_TYPE == typNUMBER::F32 && deviceProp.major >= 8 && override_enable_tf32;
    cublas_compute = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
#ifdef ENABLE_CUDNN
    create_cudnn();
#endif
/*
    printf("+-----------------------+----------------------------------------------------+\n");
    printf("| Parameter             | Value                                              |\n");
    printf("+-----------------------+----------------------------------------------------+\n");
    printf("| train data pattern    | %-50s |\n", train_data_pattern);
    printf("| val data pattern      | %-50s |\n", val_data_pattern);
    printf("| output log dir        | %-50s |\n", output_log_dir == NULL ? "NULL" : output_log_dir);
    printf("| checkpoint_every      | %-50d |\n", checkpoint_every);
    printf("| resume                | %-50d |\n", resume);
    printf("| micro batch size B    | %-50d |\n", B);
    printf("| sequence length T     | %-50d |\n", T);
    printf("| total batch size      | %-50d |\n", total_batch_size);
    printf("| LR scheduler          | %-50s |\n", lr_scheduler_type);
    printf("| learning rate (LR)    | %-50e |\n", learning_rate);
    printf("| warmup iterations     | %-50d |\n", warmup_iterations);
    printf("| final LR fraction     | %-50e |\n", final_learning_rate_frac);
    printf("| weight decay          | %-50e |\n", weight_decay);
    printf("| skip update lossz     | %-50f |\n", skip_update_lossz);
    printf("| skip update gradz     | %-50f |\n", skip_update_gradz);
    printf("| max_steps             | %-50d |\n", max_steps);
    printf("| val_loss_every        | %-50d |\n", val_loss_every);
    printf("| val_max_steps         | %-50d |\n", val_max_steps);
    printf("| sample_every          | %-50d |\n", sample_every);
    printf("| genT                  | %-50d |\n", genT);
    printf("| overfit_single_batch  | %-50d |\n", overfit_single_batch);
    printf("| use_master_weights    | %-50s |\n", use_master_weights ? "enabled" : "disabled");
    printf("| gelu_fusion           | %-50d |\n", gelu_fusion);
    printf("| recompute             | %-50d |\n", recompute);*/
    printf("+-----------------------+----------------------------------------------------+\n");
    const char* precision_str = (FLOAT_TYPE == typNUMBER::F32)
                              ? (cublas_compute == HIPBLAS_COMPUTE_32F_FAST_TF32 ? "TF32" : "FP32")
                              : (FLOAT_TYPE == typNUMBER::F16 ? "FP16" : "BF16");
    printf("| device                | %-50s |\n", deviceProp.name);
    // printf("| peak TFlops           | %-50.1f |\n", get_flops_promised(deviceProp.name, FLOAT_TYPE));
    printf("| precision             | %-50s |\n", precision_str);
    printf("+-----------------------+----------------------------------------------------+\n");
    // Streaming Multiprocessors (SMs) of NVIDIA GPUs are roughly analogous to the cores of CPUs. That is, SMs both execute computations and store state available for computation in registers, with associated caches. Compared to CPU cores, GPU SMs are simple, weak processors.
    hDevice->nCore = deviceProp.multiProcessorCount;
    cudaCheck(hipEventCreate(&cuStart));
    cudaCheck(hipEventCreate(&cuEnd));
    cudaCheck(hipProfilerStart());
    return true;
}

/*
    QKV = (B, T, 3, NH, HS) 
*/
int ROPE::FUSE_cuda(hGTensor QKV,bool isFX,int flag){
    if(Empty()){
        return -1;
    }
    int NH=n_head;    
    // hFish->GetBTC(B,T,C);
    floatX* devQ = ToX(QKV),*devK = devQ + C;    //*devPtrV = devPtrQ + 2 * C;

    const size_t s01 = QKV->ld(1);      //src0->nb[1] / ggml_type_size(src0->type);
    const size_t s02 = QKV->ld(2);      //src0->nb[2] / ggml_type_size(src0->type);
    const int n_dims     = n_rot;
    float freq_base=10000.0,freq_scale=1,ext_factor=0,attn_factor=1,beta_fast=32,beta_slow=1;
    const int32_t * pos = nullptr;  //(const int32_t *) src1_d;
    const float * freq_factors = nullptr;
    rope_corr_dims corr_dims;
    // ggml_rope_yarn_corr_dims(n_dims, n_ctx_orig, freq_base, beta_fast, beta_slow, corr_dims.v);
    floatX* dst = ToX(out);
    if(isForward()){
        grid_size = CEIL_DIV(B*T*C/2, block_size);
        // encoder_forward(ToX(cur), samps, ToX(wSrc), nullptr, 1, T, C, main_stream);
        CU_rope_<<<grid_size, block_size, 0, main_stream>>>(devQ,devQ, q_dim, head_dim, theta, n_rot,B,T,C);   
        CU_rope_<<<grid_size, block_size, 0, main_stream>>>(devK,devK, kv_dim, head_dim, theta, n_rot,B,T,C);   
    }else{
        
    }

    return 0x0;
}

//
hGTensor SelfAttention::FUSE_cuda(hGTensor inpL,hGTensor residual,hGTensor deltaIn,float* scratchF,int flag){    
    int NH=n_head;
    floatX *qkvr=ToX(Q.out);    
    float *l_att = TO<float>(trans); //(float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
    if(isForward()){    //  data=ToX(QKV->norm.out)
        hGTensor QKV=remater_qkv?GTensor::tmpFF1:Q.out;
        if(fuseNorm==nullptr){
            inpL=norm.FUSE_cuda(inpL);       
        }        
 
#ifdef ENABLE_CUDNN
        Q.Forw(QKV,inpL);  
        rope.FUSE_cuda(QKV);        
        attention_forward_cudnn(ToX(attn), l_att, ToX(QKV), B, T, NH, C_qkv, main_stream);
#else
        // if (T != model->seq_len) { // unused parts of attention buffer must be zeroed (T-dependent)
        //     cudaCheck(hipMemset(l_att, 0, B * NH * T * T * sizeof(floatX)));
        // }
        hGTensor scrath = GTensor::bt4c;        //only forward
        Q.Forw(QKV,inpL);          
        rope.FUSE_cuda(scrath); 
        attention_forward(ToX(attn), qkvr, ToX(trans), ToX(QKV), B, T, C, NH, main_stream);  //  l_atty, l_qkvr, l_att, scratch
#endif
        PrintTensor<floatX>("l_atty",ToX(attn),true,B,T,C);
        // floatX *pw=ToX(proj_cat.w), *pb=ToX0(proj_cat.b),*scratch = ToX(GTensor::scratch); //*ouput=(floatX *)out->data;

        proj_cat.Forw(GTensor::scratch,attn);   //fuMM(scratch, ToX(attn), pw, pb, B, T, C, C, main_stream);       
                
        // fused_residual_forward5(ouput, normed,mean,rstd, residual, scratch, ToX(fuseNorm->w), ToX0(fuseNorm->b), B*T, C, main_stream);
        residual_forward(ToX(out), ToX(residual), ToX(GTensor::scratch), B*T*C, main_stream);
        if(fuseNorm!=nullptr){
            float *mean=TO<float>(fuseNorm->mean),*rstd=TO<float>(fuseNorm->rstd);
            layernorm_forward(ToX(fuseNorm->out), mean, rstd, ToX(out),ToX(fuseNorm->w), ToX0(fuseNorm->b), B*T, 1, C, main_stream);
        }           
    }else{
        assert(delta!=nullptr);
        proj_cat.Back(deltaCat,attn,deltaIn,nullptr,scratchF);

        hGensor delta_attn = GTensor::bt4c; //* dl_bt4c = ToX(GTensor::bt4c); 
        if(remater_qkv)  {   
            qkvr=ToX(GTensor::tmpFF1);
            //  tmpFF1 = inpL*Q.w+Q.b
            Q.Forw(GTensor::tmpFF1,inpL); // fuMM(qkvr, data, weight, bias, B, T, C, 3*C, main_stream);
        }
#ifdef ENABLE_CUDNN
        attention_backward_cudnn(ToX(delta_attn), ToX(deltaCat), qkvr, ToX(attn), l_att, B, T, NH, C_qkv, main_stream);
#else
        assert(0);
#endif
        PrintTensor<floatX>("back of attn",ToX(delta_attn),true,B,T,C);
        // if(model->recompute >= 2) {
        //     layernorm_forward(l_ln1, l_ln1_mean, l_ln1_rstd, residual, l_ln1w, l_ln1b, B, T, C, main_stream);
        // }
        // Q.FUSE_cuda()
        matmul_backward(ToX(deltaCat), ToG(Q.w), ToG0(Q.b), ToX(delta_attn), ToX(norm.out), ToX(Q.w), scratchF, B, T, C_qkv, 3 * C_qkv, main_stream);
        // layernorm backward does += to dresidual, so it correctly accumulates gradient for the Attention block above
        float *_mean = norm.mean==nullptr?nullptr : TO<float>(norm.mean);
        norm.FUSE_cuda(residual,scratchF,deltaCat);
        // layernorm_backward(ToX(delta), ToG(norm.w), gNb, scratchF, ToX(tmpDelta), ToX(residual), ToX(norm.w), _mean, TO<float>(norm.rstd), B, T, C, main_stream);
    }
    return out;
}