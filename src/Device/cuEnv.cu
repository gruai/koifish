#include "hip/hip_runtime.h"
#include "../CLI_params.hpp"
#include "../ggex/GTensor.hpp"
#include "../g_stddef.hpp" 
// #include "../ggex/GG_util.hpp"       //ugly  "__builtin_ia32_ldtilecfg" is undefined
#include "../kGPT/llmc/cuda_common.h"
#include "../kGPT/llmc/cublas_common.h"
#include "../kGPT/llmc/matmul.cuh"
#include "../kGPT/llmc/layernorm.cuh"
#include "../kGPT/llmc/encoder.cuh"
#include "../kGPT/llmc/fused_classifier.cuh"
#include "../Manifold/Neuron.hpp"
// #include "../kGPT/llmc/mfu.h"
#define NOMINMAX
#include <cudnn_frontend.h>
namespace fe = cudnn_frontend;
// Specific configurations based on the enabled precision
#if defined(ENABLE_FP32)
    static_assert(false, "cuDNN is not supported in FP32 mode.")
    // use fp16 (note: this may require gradient scaler, currently not implemented!)
#elif defined(ENABLE_FP16)
    #define CUDNN_16BIT fe::DataType_t::HALF
#else // Default to bfloat16
    #define CUDNN_16BIT fe::DataType_t::BFLOAT16
#endif
hipblasComputeType_t cublas_compute = HIPBLAS_COMPUTE_32F;
const size_t cublaslt_workspace_size = 32 * 1024 * 1024;
hipblasLtHandle_t cublaslt_handle;
void* cublaslt_workspace = NULL;
hipStream_t main_stream=nullptr;
hipDeviceProp_t deviceProp;
static hipdnnHandle_t cudnn_handle;
static size_t cudnn_workspace_size = 0; // dynamically allocated as needed (up to 256MiB!)
static void* cudnn_workspace = NULL;
static hipEvent_t cuStart, cuEnd;

static void cuDNNCheck(hipdnnStatus_t error, const char *file, int line) {
    if (error != HIPDNN_STATUS_SUCCESS) {
        printf("[CUDNN ERROR] at file %s:%d:\n%s\n", file, line, hipdnnGetErrorString(error));
        exit(EXIT_FAILURE);
    }
};
#define cuDNNCheck(err) (cuDNNCheck(err, __FILE__, __LINE__))

static void checkCudnnFE(const fe::error_object& e, const char *file, int line) {
    if(!e.is_good()) {
        printf("[CUDNN ERROR] at file %s:%d:\n%s\n", file, line, e.err_msg.c_str());
        exit(EXIT_FAILURE);
    }
}
#define checkCudnnFE(err) checkCudnnFE(err, __FILE__, __LINE__)

enum UIDs {
    Q_UID,
    K_UID,
    V_UID,
    Attn_scale_UID,
    O_UID,
    Stats_UID,
    dO_UID,
    dQ_UID,
    dK_UID,
    dV_UID
};

// Need a cache because graph->build_operation_graph() is slow but everything else seems fast
using cache_type_fwd = std::map<std::tuple<int,int,int,int, int>, std::shared_ptr<fe::graph::Graph>>;
using cache_type_bwd = std::map<std::tuple<int,int,int,int>, std::shared_ptr<fe::graph::Graph>>;

// Loosely based on cuDNN frontend samples functions and massively simplified
auto lookup_cache_or_build_graph_fwd(int B,int H,int T,int HS, int is_inference_only) {

    static cache_type_fwd user_maintained_cache_fwd;

    auto key = std::make_tuple(B, H, T, HS, is_inference_only);

    auto it = user_maintained_cache_fwd.find(key);
    if (it != user_maintained_cache_fwd.end()) {
        return it->second;
    }

    auto graph = std::make_shared<fe::graph::Graph>();
    graph->set_io_data_type(CUDNN_16BIT)
          .set_intermediate_data_type(fe::DataType_t::FLOAT)
          .set_compute_data_type(fe::DataType_t::FLOAT);

    // QKV is (B, T, 3, NH, HS) which cuDNN can handle directly without an external permute
    auto Q = graph->tensor(fe::graph::Tensor_attributes().set_name("Q")
                               .set_dim({B, H, T, HS})
                               .set_uid(Q_UID)
                               .set_stride({3 * H * HS * T,  HS, 3 * H * HS, 1}));
    auto K = graph->tensor(fe::graph::Tensor_attributes().set_name("K")
                               .set_dim({B, H, T, HS})
                               .set_uid(K_UID)
                               .set_stride({3 * H * HS * T, HS, 3 * H * HS, 1}));
    auto V = graph->tensor(fe::graph::Tensor_attributes().set_name("V")
                               .set_dim({B, H, T, HS})
                               .set_uid(V_UID)
                               .set_stride({3 * H * HS * T, HS, 3 * H * HS, 1}));
    auto attn_scale = graph->tensor(fe::graph::Tensor_attributes().set_name("attn_scale")
                               .set_dim({1, 1, 1, 1})
                               .set_stride({1, 1, 1, 1})
                               .set_uid(Attn_scale_UID)
                               .set_is_pass_by_value(true)
                               .set_data_type(fe::DataType_t::FLOAT));

    auto sdpa_options = fe::graph::SDPA_attributes().set_name("flash_attention");
    sdpa_options.set_is_inference(is_inference_only);
    sdpa_options.set_attn_scale(attn_scale);
    sdpa_options.set_causal_mask(true);

    // Create the graph operation and get the output tensors back
    auto [O, stats] = graph->sdpa(Q, K, V, sdpa_options);

    // Output is (B, T, NH, HS) BF16/FP16 and stats for backward pass is (B, NH, T) FP32
    O->set_output(true).set_dim({B, H, T, HS}).set_stride({H * HS * T, HS, H * HS, 1}).set_uid(O_UID);

    assert(stats == nullptr || is_inference_only == false);
    if (is_inference_only == false) {
        stats->set_output(true).set_data_type(fe::DataType_t::FLOAT)
                               .set_dim({B, H, T, 1})
                               .set_stride({H * T, T, 1, 1})
                               .set_uid(Stats_UID);
    }

    checkCudnnFE(graph->validate());

    // Build the operation graph and execution part (this is the VERY SLOW PART)
    checkCudnnFE(graph->build_operation_graph(cudnn_handle));
    auto plans = graph->create_execution_plans({fe::HeurMode_t::A});
    checkCudnnFE(graph->check_support(cudnn_handle));
    checkCudnnFE(graph->build_plans(cudnn_handle));
    // Reallocate the workspace if the required size is greater than the current workspace
    // In H100 this may be around 16B
    if (graph->get_workspace_size() > cudnn_workspace_size) {
        if (cudnn_workspace_size > 0) {
            cudaCheck(hipFree(cudnn_workspace));
        }
        cudnn_workspace_size = graph->get_workspace_size();
        cudaCheck(hipMalloc(&cudnn_workspace, cudnn_workspace_size));
    }

    user_maintained_cache_fwd.insert({key, graph});

    return graph;
}

auto lookup_cache_or_build_graph_bwd(int B, int NH, int T, int HS) {
    static cache_type_bwd user_maintained_cache_bwd;

    auto key = std::make_tuple(B, NH, T, HS);

    auto it = user_maintained_cache_bwd.find(key);
    if (it != user_maintained_cache_bwd.end()) {
        return it->second;
    }

    auto graph = std::make_shared<fe::graph::Graph>();
    graph->set_io_data_type(CUDNN_16BIT)
          .set_intermediate_data_type(fe::DataType_t::FLOAT)
          .set_compute_data_type(fe::DataType_t::FLOAT);

    // (B, N, 3, NH, HS)
    // must come from inp (which means we also need to convert THAT to FP16)
    auto Q = graph->tensor(fe::graph::Tensor_attributes().set_name("Q")
                            .set_dim({B, NH, T, HS})
                            .set_uid(Q_UID)
                            .set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}));
    auto K = graph->tensor(fe::graph::Tensor_attributes().set_name("K")
                            .set_dim({B, NH, T, HS})
                            .set_uid(K_UID)
                            .set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}));
    auto V = graph->tensor(fe::graph::Tensor_attributes().set_name("V")
                            .set_dim({B, NH, T, HS})
                            .set_uid(V_UID)
                            .set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}));
    auto O = graph->tensor(fe::graph::Tensor_attributes().set_name("O")
                            .set_dim({B, NH, T, HS})
                            .set_uid(O_UID)
                            .set_stride({NH * HS * T, HS, NH * HS, 1}));
    auto dO = graph->tensor(fe::graph::Tensor_attributes().set_name("dO")
                            .set_dim({B, NH, T, HS})
                            .set_uid(dO_UID)
                            .set_stride({NH * HS * T, HS, NH * HS, 1}));

    auto stats = graph->tensor(fe::graph::Tensor_attributes().set_name("stats")
                            .set_dim({B, NH, T, 1})
                            .set_uid(Stats_UID)
                            .set_stride({NH * T, T, 1, 1})
                            .set_data_type(fe::DataType_t::FLOAT));
    auto attn_scale = graph->tensor(fe::graph::Tensor_attributes().set_name("attn_scale")
                            .set_dim({1, 1, 1, 1})
                            .set_stride({1, 1, 1, 1})
                            .set_is_pass_by_value(true)
                            .set_uid(Attn_scale_UID)
                            .set_data_type(fe::DataType_t::FLOAT));
    auto sdpa_backward_options = fe::graph::SDPA_backward_attributes().set_name("flash_attention_backward")
#if CUDNN_FRONTEND_MAJOR_VERSION > 1 || CUDNN_FRONTEND_MINOR_VERSION >= 5
                            .set_deterministic_algorithm(true) // 1.5+ needs this for determinism
#endif
                            .set_causal_mask(true)
                            .set_attn_scale(attn_scale);

    // Create the graph operation and get the output tensors back
    auto [dQ, dK, dV] = graph->sdpa_backward(Q, K, V, O, dO, stats, sdpa_backward_options);

    dQ->set_output(true).set_dim({B, NH, T, HS}).set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}).set_uid(dQ_UID);
    dK->set_output(true).set_dim({B, NH, T, HS}).set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}).set_uid(dK_UID);
    dV->set_output(true).set_dim({B, NH, T, HS}).set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}).set_uid(dV_UID);

    checkCudnnFE(graph->validate());

    // Build the operation graph and execution part (this is the VERY SLOW PART)
    checkCudnnFE(graph->build_operation_graph(cudnn_handle));
    auto plans = graph->create_execution_plans({fe::HeurMode_t::A});
    checkCudnnFE(graph->check_support(cudnn_handle));
    checkCudnnFE(graph->build_plans(cudnn_handle));

    // Reallocate the workspace if the required size is greater than the current workspace
    // By default, cuDNN uses up to 256MiB of workspace, so we don't want to just allocate the maximum
    if (graph->get_workspace_size() > cudnn_workspace_size) {
        if (cudnn_workspace_size > 0) {
            cudaCheck(hipFree(cudnn_workspace));
        }
        cudnn_workspace_size = graph->get_workspace_size();
        cudaCheck(hipMalloc(&cudnn_workspace, cudnn_workspace_size));
    }

    user_maintained_cache_bwd.insert({key, graph});
    return graph;
}

void attention_forward_cudnn(floatX* out,  // output: (B, T, NH, HS)
                             float* stats, // output for backward pass: (B, NH, T)
                             floatX* inp,  // input: (B, T, 3, NH, HS) QKV
                             int B, int T, int NH, int C, hipStream_t stream) {
    NVTX_RANGE_FN();
    int HS = C / NH; // number of features per head
    bool is_inference_only = (stats == nullptr);

    cuDNNCheck(hipdnnSetStream(cudnn_handle, stream));

    // Get graph and tensors from cache (or generate it on first use)
    auto graph = lookup_cache_or_build_graph_fwd(B, NH, T, HS, is_inference_only);

    // Prepare all the tensor pointers for executing the graph
    void* devPtrQ = inp;
    void* devPtrK = (inp + C);
    void* devPtrV = (inp + 2 * C);
    float attn_scale_cpu = 1.0 / sqrtf(HS);
    void* devPtrO = out;

    // Build variant pack
    std::unordered_map<int64_t , void*> variant_pack = {
        {Q_UID, devPtrQ}, {K_UID, devPtrK}, {V_UID, devPtrV}, {Attn_scale_UID, &attn_scale_cpu}, {O_UID, devPtrO}};

    // Add the stats tensor unless we are only doing inference (only needed for backward pass)
    if (is_inference_only == false) {
        variant_pack[Stats_UID] = stats;
    }

    // Execute graph
    checkCudnnFE(graph->execute(cudnn_handle, variant_pack, cudnn_workspace));
    cudaCheck(hipGetLastError());
}

void attention_backward_cudnn(floatX* dqkvr,                                       // output
                              floatX* dout, floatX* qkvr, floatX* o, float* stats, // inputs
                              int B, int T, int NH, int C, hipStream_t stream) {
    NVTX_RANGE_FN();
    int HS = C / NH; // number of features per head

    // Get graph and tensors from cache (or generate it on first use)
    auto graph = lookup_cache_or_build_graph_bwd(B, NH, T, HS);

    // Prepare all the tensor pointers for executing the graph
    void* devPtrQ = qkvr;
    void* devPtrK = (qkvr + NH * HS);
    void* devPtrV = (qkvr + 2 * NH * HS);
    void* devPtrO = o;
    void* devPtrdO = dout;
    void* devPtrStats = stats;
    float attn_scale_cpu = 1.0 / sqrtf(HS);

    void* devPtrdQ = dqkvr;
    void* devPtrdK = (dqkvr + NH * HS);
    void* devPtrdV = (dqkvr + 2 * NH * HS);

    // Build variant pack that links each tensor to its data pointer
    std::unordered_map<int64_t, void*> variant_pack = {
        {Q_UID, devPtrQ}, {K_UID, devPtrK}, {V_UID, devPtrV}, {O_UID, devPtrO}, {dO_UID, devPtrdO}, {Stats_UID, devPtrStats},
        {dQ_UID, devPtrdQ}, {dK_UID, devPtrdK}, {dV_UID, devPtrdV},
        {Attn_scale_UID, &attn_scale_cpu}};

    // Execute graph
    cuDNNCheck(hipdnnSetStream(cudnn_handle, stream));
    checkCudnnFE(graph->execute(cudnn_handle, variant_pack, cudnn_workspace));
    cudaCheck(hipGetLastError());
}

void create_cudnn() {
    cuDNNCheck(hipdnnCreate(&cudnn_handle));
}

void destroy_cudnn() {
    if (cudnn_workspace != NULL) { cudaCheck(hipFree(cudnn_workspace)); }
    cuDNNCheck(hipdnnDestroy(cudnn_handle));
}

bool InitCUDNN(const CLI_params&hparams,int flag){
    //  hipDriverGetVersion
    //  hipRuntimeGetVersion
    int local_device_idx = 0, override_enable_tf32 = 1;
    hipError_t err = hipSetDevice(0);
    if (err != hipSuccess) {
        printf("[InitCUDNN] failed at hipSetDevice! ERR=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    cudaCheck(hipGetDeviceProperties(&deviceProp, local_device_idx));
    if (1) {
        printf("[System]\n");
        printf("Device %d: %s\n", local_device_idx, deviceProp.name);
    }

    // set up the cuda streams. atm everything is on the single main stream
    cudaCheck(hipStreamCreate(&main_stream));
    nvtxNameCudaStreamA(main_stream, "main stream");

    // set up cuBLAS and cuBLASLt
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    bool enable_tf32 = PRECISION_MODE == PRECISION_FP32 && deviceProp.major >= 8 && override_enable_tf32;
    cublas_compute = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;

    create_cudnn();
/*
    printf("+-----------------------+----------------------------------------------------+\n");
    printf("| Parameter             | Value                                              |\n");
    printf("+-----------------------+----------------------------------------------------+\n");
    printf("| train data pattern    | %-50s |\n", train_data_pattern);
    printf("| val data pattern      | %-50s |\n", val_data_pattern);
    printf("| output log dir        | %-50s |\n", output_log_dir == NULL ? "NULL" : output_log_dir);
    printf("| checkpoint_every      | %-50d |\n", checkpoint_every);
    printf("| resume                | %-50d |\n", resume);
    printf("| micro batch size B    | %-50d |\n", B);
    printf("| sequence length T     | %-50d |\n", T);
    printf("| total batch size      | %-50d |\n", total_batch_size);
    printf("| LR scheduler          | %-50s |\n", lr_scheduler_type);
    printf("| learning rate (LR)    | %-50e |\n", learning_rate);
    printf("| warmup iterations     | %-50d |\n", warmup_iterations);
    printf("| final LR fraction     | %-50e |\n", final_learning_rate_frac);
    printf("| weight decay          | %-50e |\n", weight_decay);
    printf("| skip update lossz     | %-50f |\n", skip_update_lossz);
    printf("| skip update gradz     | %-50f |\n", skip_update_gradz);
    printf("| max_steps             | %-50d |\n", max_steps);
    printf("| val_loss_every        | %-50d |\n", val_loss_every);
    printf("| val_max_steps         | %-50d |\n", val_max_steps);
    printf("| sample_every          | %-50d |\n", sample_every);
    printf("| genT                  | %-50d |\n", genT);
    printf("| overfit_single_batch  | %-50d |\n", overfit_single_batch);
    printf("| use_master_weights    | %-50s |\n", use_master_weights ? "enabled" : "disabled");
    printf("| gelu_fusion           | %-50d |\n", gelu_fusion);
    printf("| recompute             | %-50d |\n", recompute);*/
    printf("+-----------------------+----------------------------------------------------+\n");
    const char* precision_str = (PRECISION_MODE == PRECISION_FP32)
                              ? (cublas_compute == HIPBLAS_COMPUTE_32F_FAST_TF32 ? "TF32" : "FP32")
                              : (PRECISION_MODE == PRECISION_FP16 ? "FP16" : "BF16");
    printf("| device                | %-50s |\n", deviceProp.name);
    // printf("| peak TFlops           | %-50.1f |\n", get_flops_promised(deviceProp.name, PRECISION_MODE));
    printf("| precision             | %-50s |\n", precision_str);
    printf("+-----------------------+----------------------------------------------------+\n");

    cudaCheck(hipEventCreate(&cuStart));
    cudaCheck(hipEventCreate(&cuEnd));
    cudaCheck(hipProfilerStart());
    return true;
}

hGTensor cuTensor::_Multiply(const hGTensor& b) {
    cuTensor *cuB=dynamic_cast<cuTensor *>(b.get());
    assert(cuB!=nullptr);
    // floatX* out = nullptr;
    // matmul_forward_cublaslt((floatX*)out, (floatX*)data, (floatX*)b->data, NULL, B, T, C, Vp, main_stream);
    return nullptr;
}

/*
    1. residual_forward(residual, inp1, inp2, N*C, stream);
    2. layernorm_forward(normed, mean, rstd, residual, weight, bias, N, 1, C, stream);
*/
int FUSE_ResiNormal(hGTensor hOut,hGTensor hInp1,hGTensor hInp2,hGTensor hNormed,hGTensor N_mean,hGTensor N_rstd,hGTensor w,hGTensor b,int flag){
    assert(b!=nullptr);     //I would fix this bug
    assert(hOut->isSameShape(hInp1) && hOut->isSameShape(hInp2));
    int B=GTensor::B,T=GTensor::T,C=GTensor::C;
    floatX *inp1=(floatX*)hInp1->data,*inp2=(floatX*)hInp2->data,*normed=(floatX*)hNormed->data,*residual=(floatX*)hOut->data;
    assert(N_mean->type==GGML_TYPE_F32 && N_rstd->type==GGML_TYPE_F32);
    float *mean=(float*)N_mean->data,*rstd=(float*)N_rstd->data;
    if(flag==1)
        fused_residual_forward5((floatX*)hOut->data, normed, mean,rstd, inp1, inp2, (floatX*)w->data, (floatX*)b->data, B*T, C, main_stream);
    else{
        residual_forward(residual, inp1, inp2, B*T*C, main_stream);
        layernorm_forward(normed, mean, rstd, residual, (floatX*)w->data, (floatX*)b->data, B*T, 1, C, main_stream);        
    }

    return 0x0;
}

int FUSE_QKV(hGTensor hOut,hGTensor hIn,hGTensor hQKV,hGTensor hATTN,hGTensor w,hGTensor b,int NH,hGTensor proj_w,hGTensor proj_b,int flag) {
    // float* l_att = (float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
    //floatX*out,floatX* inp, floatX* weight, floatX* bias,int B, int T, int C, int OC, hipStream_t stream,floatX* pre_gelu=NULL, int gelu_fusion=1
    int B=GTensor::B,T=GTensor::T,C=GTensor::C;
    floatX *weight=(floatX*)w->data, *bias=b==nullptr?nullptr:(floatX*)b->data;
    floatX *out=(floatX*)hOut->data,*qkv=(floatX*)hQKV->data,*attn=(floatX*)hATTN->data;
    floatX *data=(floatX*)hIn->data;
    float *stats=nullptr;
    matmul_forward_cublaslt(qkv, (floatX *)(data), weight, bias, B, T, C, 3*C, main_stream);
    //  floatX* out,  float* stats, floatX* inp,  int B, int T, int NH, int C, hipStream_t stream
    attention_forward_cudnn(attn, stats, qkv, B, T, NH, C, main_stream);

    floatX *pw=(floatX*)proj_w->data, *pb=proj_b==nullptr?nullptr:(floatX*)proj_b->data;
    matmul_forward_cublaslt(out, attn, pw, pb, B, T, C, C, main_stream);
    
    return 0x0;
}  

//
int SelfAttention::FUSE_cuda(hGTensor inpL,floatX* residual,LayerNormal*norm2,float* scratchF,int flag){
    int B=GTensor::B,T=GTensor::T,C=GTensor::C,NH=n_head;
    floatX *weight=ToX(Q.w), *bias=Q.b==nullptr?nullptr:ToX(Q.b),*qkvr=ToX(Q.out);
    floatX *inp1=ToX(proj_cat.out),*atty=ToX(attn),*data=ToX(inpL);    
    float *l_att = TO<float>(trans); //(float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
    if(isForward()){    //  data=ToX(QKV->norm.out)
        if(remater_qkv)  {
            qkvr=ToX(GTensor::scratch_ff1);
        }
#ifdef ENABLE_CUDNN        
        matmul_forward_cublaslt(qkvr, data, weight, bias, B, T, C, 3*C, main_stream);
        // PrintTensor<floatX>("l_qkvw",l_qkvw,true,3*C,C);       PrintTensor<floatX>("l_qkvb",l_qkvb,true,3*C,1);
        // PrintTensor<floatX>("l_qkvr",l_qkvr,true,B,T,3*C);
        attention_forward_cudnn(atty, l_att, qkvr, B, T, NH, C, main_stream);
#else
       floatX* l_att = ToX(QKV->trans);  //floatX* l_att = acts.att + l * B * NH * T * T;
        if (T != model->seq_len) { // unused parts of attention buffer must be zeroed (T-dependent)
            cudaCheck(hipMemset(l_att, 0, B * NH * T * T * sizeof(floatX)));
        }
        // these are only needed as scratchpads for the forward pass, but
        // need not be stored for backward
        matmul_forward_cublaslt(scratch, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C, main_stream);
        attention_forward(l_atty, l_qkvr, l_att, scratch, B, T, C, NH, main_stream);
        
#endif
        PrintTensor<floatX>("l_atty",atty,true,B,T,C);
        floatX *pw=ToX(proj_cat.w), *pb=proj_cat.b==nullptr?nullptr:ToX(proj_cat.b);
        floatX* scratch = ToX(GTensor::scratch_output),*normed=ToX(norm2->out),*ouput=(floatX *)out->data;
        matmul_forward_cublaslt(scratch, atty, pw, pb, B, T, C, C, main_stream);       
        float *mean=TO<float>(norm2->mean),*rstd=TO<float>(norm2->rstd);
        if(flag==0)
            fused_residual_forward5(ouput, normed,mean,rstd, residual, scratch, ToX(norm2->w), ToX(norm2->b), B*T, C, main_stream);
        else{
            assert(0);
            // residual_forward(ouput, inp1, inp2, B*T*C, main_stream);
            // layernorm_forward(normed,mean,rstd, inp1, ToX(norm.w), ToX(norm.b), B*T, 1, C, main_stream);        
        }
    }else{
        floatX* dl_bt4c = ToX(GTensor::scratch_bt4c),*dresidual = ToX(GTensor::scratch_btc);  
        if(remater_qkv)  {  //    data=ToX(QKV->norm.out)
            qkvr=ToX(GTensor::scratch_ff1);
            matmul_forward_cublaslt(qkvr, data, weight, bias, B, T, C, 3*C, main_stream);
        }
        attention_backward_cudnn(dl_bt4c, dl_btc, qkvr, atty, l_att, B, T, NH, C, main_stream);
        PrintTensor<floatX>("back of attn",dl_bt4c,true,B,T,C);
        // if(model->recompute >= 2) {
        //     layernorm_forward(l_ln1, l_ln1_mean, l_ln1_rstd, residual, l_ln1w, l_ln1b, B, T, C, main_stream);
        // }
        matmul_backward(dl_btc, ToG(Q.w), ToG(Q.b), dl_bt4c, ToX(norm.out), weight, scratchF, B, T, C, 3 * C, main_stream);
        // layernorm backward does += to dresidual, so it correctly accumulates gradient for the Attention block above
        layernorm_backward(dresidual, ToG(norm.w), ToG(norm.b), scratchF, dl_btc, residual, ToX(norm.w), TO<float>(norm.mean), TO<float>(norm.rstd), B, T, C, main_stream);
    }
    return 0x0;
}

//  hIn = QKV->out
int FFN::FUSE_cuda(hGTensor hIn,floatX *scratch,LayerNormal*neuron_x,int flag){
    int B=GTensor::B,T=GTensor::T,C=GTensor::C;
    floatX *ff2=ToX(down.out),*ff1=ToX(up.out),*normed=ToX(neuron_x->out);
    float *mean=TO<float>(neuron_x->mean),*rstd=TO<float>(neuron_x->rstd);
    floatX *l_fch_gelu = ToX(GTensor::scratch_output);  //  reuse the same activation buffer at each layer, as we'll re-compute the gelu during backward
    bool isBias = up.b!=nullptr;  assert(isBias);
    if(isForward()){     
        floatX * inp1_ =  ToX(norm.out);         
        if(remater_ffn)  {
            input_1 = inp1_;
            ff1=ToX(GTensor::scratch_ff1);              
        } 
        assert(ff1!=nullptr);       // ff1=gelu_forward(out, l_fch_gelu, B*T*OC, stream);
        floatX *scratch = ToX(GTensor::scratch_btc);    
        matmul_forward_cublaslt(l_fch_gelu,inp1_, (floatX*)up.w->data, (floatX*)up.b->data, B, T, C, latent, main_stream, ff1, gelu_fusion);
        // PrintTensor<floatX>("inp1",ToX(norm.out),true,B,T,C,1,-1);          PrintTensor<floatX>("ff1",ff1,true,B,T,latent,1,-1);  
        matmul_forward_cublaslt(scratch, l_fch_gelu, (floatX*)down.w->data, (floatX*)down.b->data, B, T, latent, C, main_stream);   //???
        // PrintTensor<floatX>("inp1",ToX(norm.out),true,B,T,C,1,-1);
        PrintTensor<floatX>("ffn",scratch,true,B,T,C);
        if(flag==0) {
            fused_residual_forward5(ToX(out), normed,mean,rstd, ToX(hIn), scratch, ToX(neuron_x->w), ToX(neuron_x->b), B*T, C, main_stream);
        }   else{
            assert(0);
        }
        // PrintTensor<floatX>("inp1",ToX(norm.out),true,B,T,C,1,-1);
        out->PrintX<floatX>("residual3",0,0);
    }else{
        floatX *dl_bt4c = ToX(GTensor::scratch_bt4c),*dresidual = ToX(GTensor::scratch_btc); 
        float*  scratchF = (float*) scratch;   // not the same inp1 of forward !!!
        if(input_1!=nullptr){
            input_1 =  ToX(norm.out);
            ff1=ToX(GTensor::scratch_ff1);              
            matmul_forward_cublaslt(l_fch_gelu,input_1, (floatX*)up.w->data, (floatX*)up.b->data, B, T, C, latent, main_stream, ff1, gelu_fusion);
            // norm.out->PrintX<floatX>("inp1",0,-1);          PrintTensor<floatX>("ff1",ff1,true,B,T,latent,-1);  
        }else
            gelu_forward(l_fch_gelu, ff1, B*T*latent, main_stream);  
        assert(ff1!=nullptr);   
        matmul_backward(dl_bt4c, ToG(down.w), ToG(down.b), dresidual, l_fch_gelu, ToX(down.w), scratchF, B, T, latent, C, main_stream, ff1, gelu_fusion);
        PrintTensor<floatX>("back of ffn1",dl_bt4c,true,B,T,latent);
        
        matmul_backward(residual, ToG(up.w), ToG(up.b), dl_bt4c, ToX(norm.out), ToX(up.w), scratchF, B, T, C, latent, main_stream);
        // // layernorm backward does += to the dresidual, so it correctly accumulates grad from the MLP block above
        layernorm_backward(dresidual, ToG(norm.w), ToG(norm.b), scratchF, residual, ToX(hIn), ToX(norm.w), TO<float>(norm.mean), TO<float>(norm.rstd), B, T, C, main_stream);
        matmul_backward(residual, ToG(lastQKV->proj_cat.w), ToG(lastQKV->proj_cat.b), dresidual, ToX(lastQKV->attn), ToX(lastQKV->proj_cat.w), scratchF, B, T, C, C, main_stream);
        PrintTensor<floatX>("back of ffn0",residual,true,B,T,C);
    }
    
    return 0x0;
}

int FUSE_FFN(hGTensor hOut,hGTensor hIn,hGTensor hLatent,hGTensor wUp,hGTensor bUp,hGTensor hGelu,hGTensor wDown,hGTensor bDown,int gelu_fusion,int flag) {
    int B=GTensor::B,T=GTensor::T,C=GTensor::C;
    floatX *out=(floatX*)hOut->data,*x=(floatX*)hLatent->data;
    floatX *data=(floatX*)hIn->data;
    bool isBias = bUp!=nullptr;  assert(isBias);
    matmul_forward_cublaslt(x, (floatX*)data, (floatX*)wUp->data, (floatX*)bUp->data, B, T, C, 4*C, main_stream, (floatX*)hGelu->data, gelu_fusion);
    matmul_forward_cublaslt(out, x, (floatX*)wDown->data, (floatX*)bDown->data, B, T, 4*C, C, main_stream);
    return 0x0;
}

hGTensor cuTensor::GetRow(hGTensor hOut,hGTensor token,hGTensor pos,int flag)   {
    floatX *out=(floatX*)(hOut->data),*wte=(floatX*)(data),*wpe=pos==nullptr?nullptr : (floatX*)(pos->data);
    // int nCls = shape[1],i;
    const int* inp=(int*)(token->data);
    // assert(isInRange(inp,token->size(),0,nCls));

    encoder_forward(out, inp, wte, wpe, B, T, C, main_stream);
    // PrintTensor<floatX>("wte",params.wte,true,Vp,C);        PrintTensor<floatX>("wpe",params.wpe,true,T,C);
    // PrintTensor<int>("inputs",model->inputs,true,B,T);      PrintTensor<floatX>("GetRow",ToX(embed->out),true,B,T,C);
    return hOut;
    /*
        encoder_backward(grads.wte, grads.wpe, scratchX, model->workload_indices, model->bucket_info,
                        dresidual, model->inputs, hostInput, B, T, C, random_u32(&model->rng_state), main_stream);
    */
}

/*
    layernorm_forward(floatX* out, float* mean, float* rstd, floatX* inp, const floatX* weight, const floatX* bias,         int B, int T, int C, hipStream_t stream)
    layernorm_backwar(floatX* dinp, floatX* dweight, floatX* dbias, float* scratch,const floatX* dout, const floatX* inp, const floatX* weight, const float* mean, const float* rstd,          int B, int T, int C, hipStream_t stream)
*/
hGTensor cuTensor::Normal(hGTensor hOut,hGTensor _mean,hGTensor _rstd,hGTensor w,hGTensor b,bool isForward,int flag) {
    assert(!hOut->isEmpty());
    
    assert(b!=nullptr);     //I would fix this bug
    floatX *weight=(floatX*)(w->data),*bias=b==nullptr?nullptr:(floatX*)(b->data);    
    floatX *out=(floatX*)(hOut->data); // (B, T, C)
    if(isForward)
        layernorm_forward(out, (float*)_mean->data, (float*)_rstd->data, (floatX *)data,weight,bias, B, T, C, main_stream);
    else{
        layernorm_backward(nullptr, (floatX*)(w->grad), (floatX*)(b->grad), nullptr, nullptr,nullptr, weight, 
            (float*)_mean->data, (float*)_rstd->data, B, T, C, main_stream);
    }
    
    return hOut;
}

//void fused_classifier(Type* logits, float* cuLoss,const float dloss, const int* targets,int B, int T, int V, int P, std::bool_constant<WriteDLogits> write_dlogits, hipStream_t stream) {
//float cuTensor::FusedLoss(float dloss,hGTensor hLoss,hGTensor hTarget,hGTensor hLastLayer, hGTensor w,int V,bool isForward,int flag){
int OutCLS::FUSE_cuda(hGTensor inpL,hGTensor token_embed,int flag)   {
    int B=GTensor::B,T=GTensor::T,C=GTensor::C,V=nCls,Vp=padded_nCls, gelu_fusion=1;
    assert(proj.b==nullptr);
    mean_loss = 0.0f;
    const int *targets=(int*)(target->data);
    float* cuLoss = (float*)out->data;   
    floatX* errLogits = ToX(preLogits),*z0=ToX(inpL),*w=nullptr,*gw=nullptr,*pre_gelu=nullptr;  
    floatX* errOut = ToX(GTensor::scratch_bt4c);   //B * T * 4 * C
    if(isSymProj){
        assert(proj.w==nullptr && token_embed!=nullptr);
        w=ToX(token_embed);         gw=ToG(token_embed);
    }else{
        assert(proj.w!=nullptr);
        w=ToX(proj.w);         gw=ToG(proj.w);
    }
    if(isForward()){
        // cudaCheck(hipDeviceSynchronize());         
        // cudaCheck(hipMemset(cuLoss, 0, B*T*sizeof(float)));
        cudaCheck(hipMemset(cuLoss, 0, B*T*sizeof(float)));
        assert( target->isSameShape(out) );
        constexpr std::bool_constant<true> cuFalse;    
        for(size_t i=0;i<B;i+=dB){
            size_t off=i*T*Vp,n1=i*T,nZ=i*T*C;
            off=0;      //reduce memory
            matmul_forward_cublaslt(errLogits+off, z0+nZ, w, NULL, dB, T, C, Vp, main_stream);  //[32,1024,50304]=[32,1024,768]*[768,50304]
            fused_classifier(errLogits+off, cuLoss+n1, rLoss, targets+n1, dB, T, V, Vp, cuFalse, main_stream);        //target=[32,1024]
            if(flag!=0x1001 && gw!=nullptr && errOut!=nullptr){
                matmul_cublaslt(errOut+nZ, w, errLogits+off, NULL, C, dB*T, Vp, main_stream, false, false, 0, 0, 0, 0, false,gelu_fusion >= 2 ? pre_gelu : NULL, true);   
                matmul_cublaslt(gw, z0+nZ, errLogits+off, NULL /*dbias*/, C, Vp, dB*T, main_stream, false, true, 0, 0, 0, 0,true /* accumulate */, NULL, true);                
            }                         
        }
        // fused_classifier(errLogits, cuLoss, rLoss, targets, B, T, V, Vp, cuFalse, main_stream);        //target=[32,1024]
        cudaCheck(hipMemcpy(hostLoss, cuLoss, B * T * sizeof(float), hipMemcpyDeviceToHost));                 
        cudaCheck(hipDeviceSynchronize());
        if(flag==0x1001 && gw!=nullptr && errOut!=nullptr){            //matmul_backward(errOut, gw, NULL, errLogits, z0, w, NULL, B, T, C, Vp, main_stream);      //accumulate=true  
            matmul_cublaslt(errOut, w, errLogits, NULL, C, B*T, Vp, main_stream, false, false, 0, 0, 0, 0, false,gelu_fusion >= 2 ? pre_gelu : NULL, true);
            if (gelu_fusion < 2 && pre_gelu) {
                gelu_backward_inplace(errOut, pre_gelu, B*T*C, main_stream);
            }
            matmul_cublaslt(gw, z0, errLogits, NULL /*dbias*/, C, Vp, B*T, main_stream, false, true, 0, 0, 0, 0,true /* accumulate */, NULL, true);
        }
            
        for (int i = 0; i < B*T; i++) {
            mean_loss += hostLoss[i];
        }   
        mean_loss /= B*T;
    }else{        
        // matmul_backward(errOut, gw, NULL, errLogits, z0, w, NULL, B, T, C, Vp, main_stream);
    }
    cudaCheck(hipGetLastError());
    return 0x0;
}

// #define ENABLE_CUDNN

cuTensor::~cuTensor()  {
    Free();

}

